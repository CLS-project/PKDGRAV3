/* -*- Mode: C++; tab-width: 4; indent-tabs-mode: nil; c-basic-offset: 4 -*- */
#ifdef HAVE_CONFIG_H
#include "config.h"
#endif
/*#include <nvToolsExt.h>*/

#include "cudautil.h"

#include <assert.h>
#include <stdio.h>
#include <unistd.h>
#ifdef HAVE_SYS_PARAM_H
#include <sys/param.h> /* for MAXHOSTNAMELEN, if available */
#endif

static void *CUDA_malloc(size_t nBytes) {
    void *blk = NULL;
    hipHostMalloc( &blk, nBytes);
    return blk;
    }

static void CUDA_free(void *data) {
    hipHostFree(data);
    }

static void *CUDA_gpu_malloc(size_t nBytes) {
    void *blk = NULL;
    CUDA_CHECK(hipMalloc,(&blk, nBytes));
    return blk;
    }

static void CUDA_gpu_free(void *blk) {
    CUDA_CHECK(hipFree,(blk));
    }

void CUDA_Abort(hipError_t rc, const char *fname, const char *file, int line) {
    fprintf(stderr,"%s error %d in %s(%d)\n%s\n", fname, rc, file, line, hipGetErrorString(rc));
    exit(1);
    }

extern "C"
void *CUDA_initialize(int iCore) {
    int nDevices;

    CUDA_CHECK(hipGetDeviceCount,(&nDevices))
    if (nDevices == 0 ) return NULL;
    CUDA_CHECK(hipSetDevice,(iCore % nDevices));
    CUDACTX ctx = reinterpret_cast<CUDACTX>(malloc(sizeof(struct cuda_ctx)));
    assert(ctx!=NULL);
    ctx->nWorkQueueSize = 0;
    ctx->nWorkQueueBusy = 0;
    ctx->wqCuda = ctx->wqFree = NULL;
    ctx->nodePP = NULL;
    ctx->nodePC = NULL;

    CUDA_CHECK(hipGetDeviceProperties,(&ctx->prop,iCore % nDevices));

    return ctx;
    }

extern "C"
int CUDA_flushDone(void *vcuda) {
    CUDACTX cuda = reinterpret_cast<CUDACTX>(vcuda);
    CUDAwqNode *work, **last = &cuda->wqCuda;
    while( (work=*last) !=NULL ) {
        hipError_t rc = hipEventQuery(work->event);
        if (rc==hipSuccess) {
	    assert(work->checkFcn != NULL); /* Only one hipSuccess per customer! */
            int rc = (*work->checkFcn)(work->ctx,work);
	    assert(rc == 0);
	    work->checkFcn = NULL; /* Make sure we don't do this multiple times! */
            *last = work->next;
            work->next = cuda->wqFree;
            cuda->wqFree = work;
            --cuda->nWorkQueueBusy;
            continue;
            }
        else if (rc!=hipErrorNotReady) {
            fprintf(stderr,"hipEventQuery error %d: %s\n", rc, hipGetErrorString(rc));
            exit(1);
            }
	else if (work->startTime != 0) {
	    time_t now;
	    double seconds;
	    time(&now);
	    seconds = difftime(now,work->startTime);
	    if (seconds>=30) {
		char hostname[MAXHOSTNAMELEN];
#if defined(MAXHOSTNAMELEN) && defined(HAVE_GETHOSTNAME)
		if (gethostname(hostname,MAXHOSTNAMELEN))
#endif
		    strcpy(hostname,"unknown");
		fprintf(stderr,"%s: hipEventQuery has returned hipErrorNotReady for %f seconds\n",hostname,seconds);
		work->startTime = 0;
		}
	    }
        last = &work->next;
        }
    return cuda->wqCuda != NULL;
    }

extern "C"
int CUDA_queue(void *vcuda, void *ctx,
    int (*initWork)(void *ctx,void *work),
    int (*checkWork)(void *ctx,void *work),
    int (*doneWork)(void *ctx)) {
    CUDACTX cuda = reinterpret_cast<CUDACTX>(vcuda);
    CUDAwqNode *work;
    CUDA_flushDone(vcuda);
    if (cuda->wqFree == NULL || initWork==NULL) return 0;
    work = cuda->wqFree;
    cuda->wqFree = work->next;
    ++cuda->nWorkQueueBusy;
    if ( (*initWork)(ctx,work) ) {
	    time(&work->startTime);
	    work->ctx = ctx;
	    work->checkFcn = checkWork;
	    work->next = cuda->wqCuda;
	    cuda->wqCuda = work;
	    }
    else {
        assert(0);
        }
    return 1;
    }
extern "C"
void CUDA_SetQueueSize(void *vcuda,int cudaSize, int inCudaBufSize, int outCudaBufSize) {
    CUDACTX cuda = reinterpret_cast<CUDACTX>(vcuda);
    CUDAwqNode *work;
    int hostBufSize = inCudaBufSize > outCudaBufSize ? inCudaBufSize : outCudaBufSize;
    cuda->inCudaBufSize = inCudaBufSize;
    cuda->outCudaBufSize = outCudaBufSize;
    cuda->nWorkQueueSize = cudaSize;
    while(cudaSize--) {
        work = reinterpret_cast<CUDAwqNode *>(malloc(sizeof(CUDAwqNode)));
        work->pHostBuf = CUDA_malloc(hostBufSize);
        assert(work->pHostBuf!=NULL);
        work->pCudaBufIn = CUDA_gpu_malloc(inCudaBufSize);
        assert(work->pCudaBufIn!=NULL);
        work->pCudaBufOut = CUDA_gpu_malloc(outCudaBufSize);
        assert(work->pCudaBufOut!=NULL);
        CUDA_CHECK(hipEventCreateWithFlags,( &work->event, hipEventDisableTiming ));
        CUDA_CHECK(hipStreamCreate,( &work->stream ));
        work->ctx = NULL;
        work->checkFcn = NULL;
	work->startTime = 0;
        work->next = cuda->wqFree;
        cuda->wqFree = work;
        }
    cuda->nWorkQueueBusy = 0;
    }

extern "C"
void CUDA_finish(void *vcuda) {
    CUDACTX cuda = reinterpret_cast<CUDACTX>(vcuda);
    CUDAwqNode *work;

    while( (work=cuda->wqFree) != NULL ) {
        cuda->wqFree = work->next;
        CUDA_free(work->pHostBuf);
        CUDA_gpu_free(work->pCudaBufIn);
        CUDA_gpu_free(work->pCudaBufOut);
        free(work);
        }
    hipDeviceReset();
    free(cuda);
    }
