/* -*- Mode: C++; tab-width: 4; indent-tabs-mode: nil; c-basic-offset: 4 -*- */
#ifdef HAVE_CONFIG_H
#include "config.h"
#endif

#include "cudautil.h"

#include <assert.h>
#include <stdio.h>

extern "C"
void *CUDA_malloc(size_t nBytes) {
    void *blk = NULL;
    hipHostAlloc( &blk, nBytes, 0 );
    return blk;
    }

extern "C"
void CUDA_free(void *data) {
    hipHostFree(data);
    }

extern "C"
void *CUDA_gpu_malloc(size_t nBytes) {
    void *blk = NULL;
    CUDA_CHECK(hipMalloc,(&blk, nBytes));
    return blk;
    }

extern "C"
void CUDA_gpu_free(void *blk) {
    CUDA_CHECK(hipFree,(blk));
    }

void CUDA_Abort(hipError_t rc, const char *fname, const char *file, int line) {
    fprintf(stderr,"%s error %d in %s(%d)\n%s\n", fname, rc, file, line, hipGetErrorString(rc));
    exit(1);
    }

extern "C" void CUDAsetupPP(void);
extern "C" void CUDAsetupPC(void);
#if 0

extern "C"
void *CUDA_initialize(int iProc,int iWorkQueueSize, size_t tileSize, size_t ParticlesSize, size_t OutSize) {
    int nDevices, i;

    CUDA_CHECK(hipGetDeviceCount,(&nDevices))
    if (nDevices == 0 ) return NULL;
    CUDA_CHECK(hipSetDevice,(iProc % nDevices));
    /*CUDA_CHECK(hipSetDevice,(iProc%1));*/



    CUDACTX ctx = reinterpret_cast<CUDACTX>(malloc(sizeof(struct cuda_ctx)));
    assert(ctx!=NULL);
    ctx->iWorkQueueSize = iWorkQueueSize;
    ctx->in = NULL;
    ctx->block = NULL;

    CUDA_CHECK(hipGetDeviceProperties,(&ctx->prop,iProc % nDevices));

    for(i=0; i<iWorkQueueSize; ++i) {

        gpuInput *in = reinterpret_cast<gpuInput *>(malloc(sizeof(gpuInput)));
        assert(in!=NULL);
        in->next = ctx->in;
        ctx->in = in;
        CUDA_CHECK(hipMalloc,(&in->in, ParticlesSize));
        CUDA_CHECK(hipHostAlloc,(reinterpret_cast<void **>(&in->cpuIn), ParticlesSize, 0));
        CUDA_CHECK(hipEventCreate,( &in->event ));

        gpuBlock *blk = reinterpret_cast<gpuBlock *>(malloc(sizeof(gpuBlock)));
        assert(blk!=NULL);
        blk->next = ctx->block;
        ctx->block = blk;

        CUDA_CHECK(hipMalloc,(&blk->gpuBlk, tileSize));
        CUDA_CHECK(hipMalloc,(reinterpret_cast<void **>(&blk->gpuResults), OutSize));
        CUDA_CHECK(hipHostAlloc,(reinterpret_cast<void **>(&blk->cpuResults), OutSize, 0));
        CUDA_CHECK(hipEventCreate,( &blk->event ));
        CUDA_CHECK(hipStreamCreate,( &blk->stream ));
        }


    CUDAsetupPP();
    CUDAsetupPC();

    return ctx;
    }

extern "C"
void CUDA_finish(void *vctx) {
    CUDACTX ctx = reinterpret_cast<CUDACTX>(vctx);

    while(ctx->in != NULL) {
        gpuInput *in = ctx->in;
        ctx->in = in->next;
        hipFree(in->in);
        hipEventDestroy( in->event );
        free(in);
        }

    while(ctx->block != NULL) {
        gpuBlock *blk = ctx->block;
        ctx->block = blk->next;
        hipFree(blk->gpuBlk);
        hipFree(reinterpret_cast<void *>(blk->gpuResults));
        hipHostFree(reinterpret_cast<void *>(blk->cpuResults));
        hipEventDestroy( blk->event );
        hipStreamDestroy( blk->stream );
        free(blk);
        }

    free(ctx);
    }
#endif

extern "C"
void *CUDA_initialize(int iCore) {
    int nDevices;

    CUDA_CHECK(hipGetDeviceCount,(&nDevices))
    if (nDevices == 0 ) return NULL;
    CUDA_CHECK(hipSetDevice,(iCore % nDevices));
    CUDACTX ctx = reinterpret_cast<CUDACTX>(malloc(sizeof(struct cuda_ctx)));
    assert(ctx!=NULL);
    ctx->wqSize = 0;
    ctx->wqCuda = ctx->wqFree = NULL;

    CUDA_CHECK(hipGetDeviceProperties,(&ctx->prop,iCore % nDevices));



#if 0
    for(i=0; i<iWorkQueueSize; ++i) {

        gpuInput *in = reinterpret_cast<gpuInput *>(malloc(sizeof(gpuInput)));
        assert(in!=NULL);
        in->next = ctx->in;
        ctx->in = in;
        CUDA_CHECK(hipMalloc,(&in->in, ParticlesSize));
        CUDA_CHECK(hipHostAlloc,(reinterpret_cast<void **>(&in->cpuIn), ParticlesSize, 0));
        CUDA_CHECK(hipEventCreate,( &in->event ));

        gpuBlock *blk = reinterpret_cast<gpuBlock *>(malloc(sizeof(gpuBlock)));
        assert(blk!=NULL);
        blk->next = ctx->block;
        ctx->block = blk;

        CUDA_CHECK(hipMalloc,(&blk->gpuBlk, tileSize));
        CUDA_CHECK(hipMalloc,(reinterpret_cast<void **>(&blk->gpuResults), OutSize));
        CUDA_CHECK(hipHostAlloc,(reinterpret_cast<void **>(&blk->cpuResults), OutSize, 0));
        CUDA_CHECK(hipEventCreate,( &blk->event ));
        CUDA_CHECK(hipStreamCreate,( &blk->stream ));
        }
#endif
    return ctx;
    }

extern "C"
int CUDA_flushDone(void *vcuda) {
    CUDACTX cuda = reinterpret_cast<CUDACTX>(vcuda);
    CUDAwqNode *work, **last = &cuda->wqCuda;
    while( (work=*last) !=NULL ) {
        hipError_t rc = hipEventQuery(work->event);
        if (rc==hipSuccess) {
            if ( (*work->checkFcn)(work->ctx,work) == 0) {
                *last = work->next;
                work->next = cuda->wqFree;
                cuda->wqFree = work;
                continue;
                }
            }
        else if (rc!=hipErrorNotReady) {
            fprintf(stderr,"hipEventQuery error %d: %s\n", rc, hipGetErrorString(rc));
            exit(1);
            }
        last = &work->next;
        }
    return cuda->wqCuda != NULL;
    }

extern "C"
int CUDA_queue(void *vcuda, void *ctx,
    int (*initWork)(void *ctx,void *work),
    int (*checkWork)(void *ctx,void *work),
    int (*doneWork)(void *ctx)) {
    CUDACTX cuda = reinterpret_cast<CUDACTX>(vcuda);
    CUDAwqNode *work;
    CUDA_flushDone(vcuda);
    if (cuda->wqFree == NULL || initWork==NULL) return 0;
    work = cuda->wqFree;
    cuda->wqFree = work->next;
    if ( (*initWork)(ctx,work) ) {
	    work->ctx = ctx;
	    work->checkFcn = checkWork;
	    work->doneFcn = doneWork;
	    work->next = cuda->wqCuda;
	    cuda->wqCuda = work;
	    }
    else {
        assert(0);
        }
    return 1;
    }
extern "C"
void CUDA_SetQueueSize(void *vcuda,int cudaSize, int inCudaBufSize) {
    CUDACTX cuda = reinterpret_cast<CUDACTX>(vcuda);
    CUDAwqNode *work;
    while(cudaSize--) {
        work = reinterpret_cast<CUDAwqNode *>(malloc(sizeof(CUDAwqNode)));
        work->pHostBuf = CUDA_malloc(inCudaBufSize);
        work->pCudaBuf = CUDA_gpu_malloc(inCudaBufSize);
        CUDA_CHECK(hipEventCreate,( &work->event ));
        CUDA_CHECK(hipStreamCreate,( &work->stream ));
        work->ctx = NULL;
        work->checkFcn = NULL;
        work->doneFcn = NULL;
        work->next = cuda->wqFree;
        cuda->wqFree = work;
//        ++mdl->wqCudaMaxSize;
        }
    }
