#include "hip/hip_runtime.h"
/* -*- Mode: C++; tab-width: 8; indent-tabs-mode: nil; c-basic-offset: 4 -*- */
#ifdef HAVE_CONFIG_H
#include "config.h"
#endif
#include <stdio.h>
#include "cudautil.h"

#define SYNC_RATE 16  // Must be: 1, 2, 4, 8, 16
#define SYNC_MASK (SYNC_RATE-1)
#define WARPS 4
#define WIDTH ILP_PART_PER_BLK

#include "ilp.h"
#include "ilc.h"
#include "basetype.h"

/*
** The following are the basically the same as ILP_BLK and ILC_BLK,
** but we need to be able to alter their sizes.
*/

template<int n>
struct ilpBlk {
    float dx[n], dy[n], dz[n];    /* Offset from ilp->cx, cy, cz */
    float m[n];             /* Mass */
    float fourh2[n];        /* Softening: calculated */
    };

template<int n>
struct ilcBlk {
    float dx[n],dy[n],dz[n];
    float xxxx[n],xxxy[n],xxxz[n],xxyz[n],xxyy[n],yyyz[n],xyyz[n],xyyy[n],yyyy[n];
    float xxx[n],xyy[n],xxy[n],yyy[n],xxz[n],yyz[n],xyz[n];
    float xx[n],xy[n],xz[n],yy[n],yz[n];
    float x[n],y[n],z[n];
    float m[n],u[n];
    };

// One of these entries for each interaction block
struct ppWorkUnit {
    int nP;   // Number of particles
    int iP;   // Index of first particle
    int nI;   // Number of interactions in the block
    int iO;   // Index of the output block
    };


struct ppInput {
    float dx, dy, dz;
    float ax, ay, az;
    float fSoft2;
    float dImaga;
    };

/* Each thread block outputs this for each particle */
struct __align__(32) ppResult {
    float ax;
    float ay;
    float az;
    float fPot;
    float dirsum;
    float normsum;
    };


CUDAwqNode *getNode(CUDACTX cuda) {
    CUDAwqNode *work;
    if (cuda->wqFree == NULL) return NULL;
    work = cuda->wqFree;
    cuda->wqFree = work->next;
    work->ctx = cuda;
    work->checkFcn = NULL;
    work->next = cuda->wqCuda;
    work->ppSizeIn = 0;
    work->ppSizeOut = 0;
    work->ppnBuffered = 0;
    work->ppnBlocks = 0;
    return work;
    }

#define NP_ALIGN (128/sizeof(ppResult))
#define NP_ALIGN_MASK (NP_ALIGN-1)


/*
** Occupancy (theoretical):
**   Compute 2.x:  8 (blocks) * PP_THREADS (128) = 1024 (2/3 MAX)
**   Compute 3.x: 16 (blocks) * PP_THREADS (128) = 2048 (MAX!)
**
** To reduce memory, we syncthreads() and flush the particles
** results every SYNC_RATE particles => 8 seems a good choice.
**1
** Shared memory requirements
**  - Particles         32 * nSyncRate (8)                 =  256
**  - warp reduction     4 * nWarps (4) * 32 (threads)     =  512
**  - warp results      24 * nSyncRate (8)  * nWarps (4)   =  768
** TOTAL 1536 * 16 blocks = 24 KB
** Same nSyncRate=16: 2560 * 16 blocks = 40 KB
**
** nvcc -DHAVE_CONFIG_H --ptxas-options=-v -c  -I. -arch=sm_20 cudapp.cu
** ptxas info    : 11 bytes gmem, 8 bytes cmem[14]
** ptxas info    : Compiling entry function '_Z6cudaPPILi4ELi16EEvPK10ppWorkUnitPK7ppInputPK7ILP_BLKP8ppResult' for 'sm_20'
** ptxas info    : Function properties for _Z6cudaPPILi4ELi16EEvPK10ppWorkUnitPK7ppInputPK7ILP_BLKP8ppResult
**     0 bytes stack frame, 0 bytes spill stores, 0 bytes spill loads
** ptxas info    : Used 35 registers, 2560 bytes smem, 64 bytes cmem[0], 12 bytes cmem[16]
**
** Interaction lists limits.
**   Compute 2.x: 65535 * PP_THREADS (128) = 8 million
**   Compute 3.x: effectively unlimited
**
*/

// A good number for nWarps is 4 giving 128 threads per thread block, nSyncRate=8
// Each thread block outputs ay,ay,az,fPot,dirsum,normsum for each particle
template <int nIntPerTB,int nWarps,int nSyncRate>
__global__ void cudaInteract(
    const ppWorkUnit * __restrict__ work,
    const ppInput * __restrict__ pPart,
    const ilpBlk<WIDTH> * __restrict__ blk,
    ppResult *out) {
    int i, iSync;

    // Calculate our interaction and particle group
    int iI = threadIdx.y*blockDim.x + threadIdx.x;
    int iWarp = iI / 32;
    int iTinW = iI % 32;
    int iWork = blockIdx.x*blockDim.z + threadIdx.z;
    int nP = work[iWork].nP; // Number of particles
    pPart += work[iWork].iP; // First particle
    int nI = work[iWork].nI; // Number of interactions
    blk += iWork*blockDim.y; // First iteraction
    out += work[iWork].iO;   // Result for each particle

    __shared__ union {
        ppInput P[nSyncRate];
        float   W[nSyncRate*sizeof(ppInput)/sizeof(float)];
        } Particles;

    __shared__ float reduce[nWarps][32];

    __shared__ float wX[nSyncRate][nWarps];
    __shared__ float wY[nSyncRate][nWarps];
    __shared__ float wZ[nSyncRate][nWarps];

    __shared__ float wPot[nSyncRate][nWarps];
    __shared__ float wDirsum[nSyncRate][nWarps];
    __shared__ float wNormsum[nSyncRate][nWarps];


    // Load the interaction. It is blocked for performance.
    float iX,iY,iZ,iM,ifourh2;
    if (iI < nI) {
        iX = blk[threadIdx.y].dx[threadIdx.x];
        iY = blk[threadIdx.y].dy[threadIdx.x];
        iZ = blk[threadIdx.y].dz[threadIdx.x];
        iM = blk[threadIdx.y].m[threadIdx.x];
        ifourh2 = blk[threadIdx.y].fourh2[threadIdx.x];
        }
    for(iSync=0; iSync<nP; iSync += nSyncRate) {
        int iEnd = nP - iSync;
        if (iEnd > nSyncRate) iEnd=nSyncRate;
        // Preload the bucket of particles - this is a memcpy
        if (iI < iEnd*sizeof(ppInput) / sizeof(float)) {
            Particles.W[iI] = (reinterpret_cast<const float *>(pPart+iSync))[iI];
            }
        if (iI < iEnd ) { 
            float ax = Particles.P[iI].ax;
            float ay = Particles.P[iI].ay;
            float az = Particles.P[iI].az;
            Particles.P[iI].dImaga = ax*ax + ay*ay + az*az;
            if (Particles.P[iI].dImaga > 0.0f) Particles.P[iI].dImaga = rsqrtf(Particles.P[iI].dImaga);
            }
        __syncthreads();

        for( i=0; i<iEnd; ++i) {
            float ax=0.0f, ay=0.0f, az=0.0f, fPot=0.0f, dirsum=0.0f, normsum=0.0f;
            if (iI < nI) {
                float fourh2,dir,dir2,dir3;
                float dx = iX + Particles.P[i].dx;
                float dy = iY + Particles.P[i].dy;
                float dz = iZ + Particles.P[i].dz;
                float d2 = dx*dx + dy*dy + dz*dz;
                if (d2 != 0.0f ) { /* Ignore self interactions */
                    fourh2 = ifourh2;
                    if (d2 > fourh2) fourh2 = d2;
                    dir = rsqrtf(fourh2);
                    dir2 = dir*dir;
                    dir3 = dir2*dir;
                    if (d2 < fourh2) {
                        /*
                        ** This uses the Dehnen K1 kernel function now, it's fast!
                        */
                        dir2 *= d2;
                        dir2 = 1.0f - dir2;
                        dir *= 1.0f + dir2*(0.5f + dir2*(3.0f/8.0f + dir2*(45.0f/32.0f)));
                        dir3 *= 1.0f + dir2*(1.5f + dir2*(135.0f/16.0f));
                        }
                    dir3 *= -iM;
                    ax = dx * dir3;
                    ay = dy * dir3;
                    az = dz * dir3;
                    fPot = -iM*dir;
                    /*
                    ** Calculations for determining the timestep.
                    */
                    float adotai;
                    adotai = Particles.P[i].ax*ax + Particles.P[i].ay*ay + Particles.P[i].az*az;
                    if (adotai > 0.0f && d2 >= Particles.P[i].fSoft2 ) {
                        adotai *= Particles.P[i].dImaga;
                        dirsum = dir*adotai*adotai;
                        normsum = adotai*adotai;
                        }
                    }
                }
            // Horizontal add within each warp -- no sychronization required
            warpReduceAndStore<float,32>(reduce[iWarp],iTinW,ax,           &wX[i][iWarp]);
            warpReduceAndStore<float,32>(reduce[iWarp],iTinW,ay,           &wY[i][iWarp]);
            warpReduceAndStore<float,32>(reduce[iWarp],iTinW,az,           &wZ[i][iWarp]);
            warpReduceAndStore<float,32>(reduce[iWarp],iTinW,fPot,       &wPot[i][iWarp]);
            warpReduceAndStore<float,32>(reduce[iWarp],iTinW,dirsum,  &wDirsum[i][iWarp]);
            warpReduceAndStore<float,32>(reduce[iWarp],iTinW,normsum,&wNormsum[i][iWarp]);
            }

        __syncthreads();
        // Assuming four warps & SYNC of 8, the cache looks like this:
        // Cache: P0 P0 P0 P0 P1 P1 P1 P1 P2 ... P6 P7 P7 P7 P7
        // every set of 4 threads does another reduce. As long as the
        // number of warps is a power of 2 <= the warp size (32), we
        // can do this step without any further synchronization.

        int nOut = iEnd * nWarps; // Normally 32
        if (iI<nOut) {
            int iP    = iI & ~(nWarps-1); // 0,4,8,...
            int iWarp = iI &  (nWarps-1); // 0 .. 3
            int iOut  = iI / nWarps + iSync;
            warpReduceAndStore<float,nWarps>(      &wX[0][0]+iP,iWarp,&out[iOut].ax);
            warpReduceAndStore<float,nWarps>(      &wY[0][0]+iP,iWarp,&out[iOut].ay);
            warpReduceAndStore<float,nWarps>(      &wZ[0][0]+iP,iWarp,&out[iOut].az);
            warpReduceAndStore<float,nWarps>(    &wPot[0][0]+iP,iWarp,&out[iOut].fPot);
            warpReduceAndStore<float,nWarps>( &wDirsum[0][0]+iP,iWarp,&out[iOut].dirsum);
            warpReduceAndStore<float,nWarps>(&wNormsum[0][0]+iP,iWarp,&out[iOut].normsum);
            }
        }
    }





template <int nIntPerTB,int nWarps,int nSyncRate>
__global__ void cudaInteract(
    const ppWorkUnit * __restrict__ work,
    const ppInput * __restrict__ pPart,
    const ilcBlk<WIDTH> * __restrict__ blk,
    ppResult *out) {
    int i, iSync;

    // Calculate our interaction and particle group
    int iI = threadIdx.y*blockDim.x + threadIdx.x;
    int iWarp = iI / 32;
    int iTinW = iI % 32;
    int iWork = blockIdx.x*blockDim.z + threadIdx.z;
    int nP = work[iWork].nP; // Number of particles
    pPart += work[iWork].iP; // First particle
    int nI = work[iWork].nI; // Number of interactions
    blk += iWork*blockDim.y; // First iteraction
    out += work[iWork].iO;   // Result for each particle

    __shared__ union {
        ppInput P[nSyncRate];
        float   W[nSyncRate*sizeof(ppInput)/sizeof(float)];
        } Particles;

    __shared__ float reduce[nWarps][32];

    __shared__ float wX[nSyncRate][nWarps];
    __shared__ float wY[nSyncRate][nWarps];
    __shared__ float wZ[nSyncRate][nWarps];

    __shared__ float wPot[nSyncRate][nWarps];
    __shared__ float wDirsum[nSyncRate][nWarps];
    __shared__ float wNormsum[nSyncRate][nWarps];


    // Load the interaction. It is blocked for performance.
    float Idx,Idy,Idz;
    float Ixxxx,Ixxxy,Ixxxz,Ixxyz,Ixxyy,Iyyyz,Ixyyz,Ixyyy,Iyyyy;
    float Ixxx,Ixyy,Ixxy,Iyyy,Ixxz,Iyyz,Ixyz;
    float Ixx,Ixy,Ixz,Iyy,Iyz;
#ifdef USE_DIAPOLE
    float Ix,Iy,Iz;
#endif
    float Im,Iu;
    if (iI < nI) {
        Idx = blk[threadIdx.y].dx[threadIdx.x];
        Idy = blk[threadIdx.y].dy[threadIdx.x];
        Idz = blk[threadIdx.y].dz[threadIdx.x];
        Ixxxx = blk[threadIdx.y].xxxx[threadIdx.x];
        Ixxxy = blk[threadIdx.y].xxxy[threadIdx.x];
        Ixxxz = blk[threadIdx.y].xxxz[threadIdx.x];
        Ixxyz = blk[threadIdx.y].xxyz[threadIdx.x];
        Ixxyy = blk[threadIdx.y].xxyy[threadIdx.x];
        Iyyyz = blk[threadIdx.y].yyyz[threadIdx.x];
        Ixyyz = blk[threadIdx.y].xyyz[threadIdx.x];
        Ixyyy = blk[threadIdx.y].xyyy[threadIdx.x];
        Iyyyy = blk[threadIdx.y].yyyy[threadIdx.x];
        Ixxx = blk[threadIdx.y].xxx[threadIdx.x];
        Ixyy = blk[threadIdx.y].xyy[threadIdx.x];
        Ixxy = blk[threadIdx.y].xxy[threadIdx.x];
        Iyyy = blk[threadIdx.y].yyy[threadIdx.x];
        Ixxz = blk[threadIdx.y].xxz[threadIdx.x];
        Iyyz = blk[threadIdx.y].yyz[threadIdx.x];
        Ixyz = blk[threadIdx.y].xyz[threadIdx.x];
        Ixx = blk[threadIdx.y].xx[threadIdx.x];
        Ixy = blk[threadIdx.y].xy[threadIdx.x];
        Ixz = blk[threadIdx.y].xz[threadIdx.x];
        Iyy = blk[threadIdx.y].yy[threadIdx.x];
        Iyz = blk[threadIdx.y].yz[threadIdx.x];
#ifdef USE_DIAPOLE
        Ix = blk[threadIdx.y].x[threadIdx.x];
        Iy = blk[threadIdx.y].y[threadIdx.x];
        Iz = blk[threadIdx.y].z[threadIdx.x];
#endif
        Im = blk[threadIdx.y].m[threadIdx.x];
        Iu = blk[threadIdx.y].u[threadIdx.x];
        }
    for(iSync=0; iSync<nP; iSync += nSyncRate) {
        int iEnd = nP - iSync;
        if (iEnd > nSyncRate) iEnd=nSyncRate;
        // Preload the bucket of particles - this is a memcpy
        if (iI < iEnd*sizeof(ppInput) / sizeof(float)) {
            Particles.W[iI] = (reinterpret_cast<const float *>(pPart+iSync))[iI];
            }
        if (iI < iEnd ) { 
            float ax = Particles.P[iI].ax;
            float ay = Particles.P[iI].ay;
            float az = Particles.P[iI].az;
            Particles.P[iI].dImaga = ax*ax + ay*ay + az*az;
            if (Particles.P[iI].dImaga > 0.0f) Particles.P[iI].dImaga = rsqrtf(Particles.P[iI].dImaga);
            }
        __syncthreads();

        for( i=0; i<iEnd; ++i) {
            float ax=0.0f, ay=0.0f, az=0.0f, fPot=0.0f, dirsum=0.0f, normsum=0.0f;
            if (iI < nI) {
		const float onethird = 1.0f/3.0f;
		float dx = Idx + Particles.P[i].dx;
		float dy = Idy + Particles.P[i].dy;
		float dz = Idz + Particles.P[i].dz;
		float d2 = dx*dx + dy*dy + dz*dz;
		float dir = rsqrtf(d2);
		float u = Iu*dir;
		float g1 = dir*u;
		float g2 = 3.0f*g1*u;
		float g3 = 5.0f*g2*u;
		float g4 = 7.0f*g3*u;
		/*
		** Calculate the funky distance terms.
		*/
		float x = dx*dir;
		float y = dy*dir;
		float z = dz*dir;
		float xx = 0.5f*x*x;
		float xy = x*y;
		float xz = x*z;
		float yy = 0.5f*y*y;
		float yz = y*z;
		float zz = 0.5f*z*z;
		float xxx = x*(onethird*xx - zz);
		float xxz = z*(xx - onethird*zz);
		float yyy = y*(onethird*yy - zz);
		float yyz = z*(yy - onethird*zz);
		xx -= zz;
		yy -= zz;
		float xxy = y*xx;
		float xyy = x*yy;
		float xyz = xy*z;

	    /*
	    ** Now calculate the interaction up to Hexadecapole order.
	    */
		float tx = g4*(Ixxxx*xxx + Ixyyy*yyy + Ixxxy*xxy + Ixxxz*xxz + Ixxyy*xyy + Ixxyz*xyz + Ixyyz*yyz);
		float ty = g4*(Ixyyy*xyy + Ixxxy*xxx + Iyyyy*yyy + Iyyyz*yyz + Ixxyy*xxy + Ixxyz*xxz + Ixyyz*xyz);
		float tz = g4*(-Ixxxx*xxz - (Ixyyy + Ixxxy)*xyz - Iyyyy*yyz + Ixxxz*xxx + Iyyyz*yyy - Ixxyy*(xxz + yyz) + Ixxyz*xxy + Ixyyz*xyy);
		g4 = 0.25f*(tx*x + ty*y + tz*z);
		xxx = g3*(Ixxx*xx + Ixyy*yy + Ixxy*xy + Ixxz*xz + Ixyz*yz);
		xxy = g3*(Ixyy*xy + Ixxy*xx + Iyyy*yy + Iyyz*yz + Ixyz*xz);
		xxz = g3*(-(Ixxx + Ixyy)*xz - (Ixxy + Iyyy)*yz + Ixxz*xx + Iyyz*yy + Ixyz*xy);
		g3 = onethird*(xxx*x + xxy*y + xxz*z);
		xx = g2*(Ixx*x + Ixy*y + Ixz*z);
		xy = g2*(Iyy*y + Ixy*x + Iyz*z);
		xz = g2*(-(Ixx + Iyy)*z + Ixz*x + Iyz*y);
		g2 = 0.5f*(xx*x + xy*y + xz*z);
		float g0 = dir * Im;
		fPot = -(g0 + g2 + g3 + g4);
		g0 += 5.0f*g2 + 7.0f*g3 + 9.0f*g4;
#ifdef USE_DIAPOLE
		yy = g1*Ix;
		yz = g1*Iy;
		zz = g1*Iz;
		g1 = yy*x + yz*y + zz*z;
		fPot -= g1;
		g0 += 3.0f*g1; 
#else
		yy = 0.0f;
		yz = 0.0f;
		zz = 0.0f;
#endif
		ax = dir*(yy + xx + xxx + tx - x*g0);
		ay = dir*(yz + xy + xxy + ty - y*g0);
		az = dir*(zz + xz + xxz + tz - z*g0);

                /*
                ** Calculations for determining the timestep.
                */
                float adotai;
                adotai = Particles.P[i].ax*ax + Particles.P[i].ay*ay + Particles.P[i].az*az;
                if (adotai > 0.0f && d2 >= Particles.P[i].fSoft2 ) {
                    adotai *= Particles.P[i].dImaga;
                    dirsum = dir*adotai*adotai;
                    normsum = adotai*adotai;
                    }
                }
            // Horizontal add within each warp -- no sychronization required
            warpReduceAndStore<float,32>(reduce[iWarp],iTinW,ax,     &wX[i][iWarp]);
            warpReduceAndStore<float,32>(reduce[iWarp],iTinW,ay,     &wY[i][iWarp]);
            warpReduceAndStore<float,32>(reduce[iWarp],iTinW,az,     &wZ[i][iWarp]);
            warpReduceAndStore<float,32>(reduce[iWarp],iTinW,fPot,   &wPot[i][iWarp]);
            warpReduceAndStore<float,32>(reduce[iWarp],iTinW,dirsum, &wDirsum[i][iWarp]);
            warpReduceAndStore<float,32>(reduce[iWarp],iTinW,normsum,&wNormsum[i][iWarp]);
	}

        __syncthreads();
        // Assuming four warps & SYNC of 8, the cache looks like this:
        // Cache: P0 P0 P0 P0 P1 P1 P1 P1 P2 ... P6 P7 P7 P7 P7
        // every set of 4 threads does another reduce. As long as the
        // number of warps is a power of 2 <= the warp size (32), we
        // can do this step without any further synchronization.

        int nOut = iEnd * nWarps; // Normally 32
        if (iI<nOut) {
            int iP    = iI & ~(nWarps-1); // 0,4,8,...
            int iWarp = iI &  (nWarps-1); // 0 .. 3
            int iOut  = iI / nWarps + iSync;
            warpReduceAndStore<float,nWarps>(      &wX[0][0]+iP,iWarp,&out[iOut].ax);
            warpReduceAndStore<float,nWarps>(      &wY[0][0]+iP,iWarp,&out[iOut].ay);
            warpReduceAndStore<float,nWarps>(      &wZ[0][0]+iP,iWarp,&out[iOut].az);
            warpReduceAndStore<float,nWarps>(    &wPot[0][0]+iP,iWarp,&out[iOut].fPot);
            warpReduceAndStore<float,nWarps>( &wDirsum[0][0]+iP,iWarp,&out[iOut].dirsum);
            warpReduceAndStore<float,nWarps>(&wNormsum[0][0]+iP,iWarp,&out[iOut].normsum);
            }
        }
    }










extern "C"
void pkdParticleWorkDone(workParticle *wp);

template<int nIntPerTB>
int CUDAcheckWorkInteraction( void *vpp, void *vwork ) {
    CUDAwqNode *work = reinterpret_cast<CUDAwqNode *>(vwork);
    ppResult *pR       = reinterpret_cast<ppResult *>(work->pHostBuf);
    int ib, iw, ip;

    for( ib=0; ib<work->ppnBuffered; ++ib) {
        workParticle *wp = work->ppWP[ib];
        PINFOOUT *pInfoOut = wp->pInfoOut;
        int nWork = (work->ppNI[ib] + nIntPerTB - 1) / nIntPerTB;
        for(iw=0; iw<nWork; ++iw) {
            for(ip=0; ip<wp->nP; ++ip) {
                pInfoOut[ip].a[0]    += pR[ip].ax;
                pInfoOut[ip].a[1]    += pR[ip].ay;
                pInfoOut[ip].a[2]    += pR[ip].az;
                pInfoOut[ip].fPot    += pR[ip].fPot;
                pInfoOut[ip].dirsum  += pR[ip].dirsum;
                pInfoOut[ip].normsum += pR[ip].normsum;
                }
            pR += wp->nP;
            }
        pkdParticleWorkDone(wp);

        }
    return 0;
    }

extern "C"
void CUDAsetupPP(void) {
    //hipFuncSetCacheConfig(reinterpret_cast<const void*>(cudaPP),hipFuncCachePreferL1);
    }


template<int nIntPerTB, typename BLK>
void CUDA_sendWork(CUDACTX cuda,CUDAwqNode **head) {
    CUDAwqNode *work = *head;
    if (work != NULL) {
        int i, j;
        int iI=0, iP=0, iO=0;
        int nBufferOut = 0;
        int nBlkPer = nIntPerTB / WIDTH;
        int nWork = work->ppnBlocks/nBlkPer;

        // The interation blocks -- already copied to the host memory
        BLK * __restrict__ blkHost = reinterpret_cast<BLK*>(work->pHostBuf);
        BLK * __restrict__ blkCuda = reinterpret_cast<BLK*>(work->pCudaBufIn);
        
        // The interaction block descriptors
        ppWorkUnit * __restrict__ wuHost = reinterpret_cast<ppWorkUnit *>(blkHost + work->ppnBlocks);
        ppWorkUnit * __restrict__ wuCuda = reinterpret_cast<ppWorkUnit *>(blkCuda + work->ppnBlocks);

        // The particle information
        ppInput * __restrict__ partHost = reinterpret_cast<ppInput *>(wuHost + ((nWork+7)&~7));
        ppInput * __restrict__ partCuda = reinterpret_cast<ppInput *>(wuCuda + ((nWork+7)&~7));

        for( i=0; i<work->ppnBuffered; ++i) {
            const int nP = work->ppWP[i]->nP;
            PINFOIN *pInfoIn = work->ppWP[i]->pInfoIn;
            int nPaligned = (nP+NP_ALIGN_MASK) & ~NP_ALIGN_MASK;
            int nInteract = work->ppNI[i];
            int nBlocks = (nInteract+nIntPerTB-1) / nIntPerTB;

            // Generate a interaction block descriptor for each block
            for(j=0; j<nBlocks; ++j) {
                wuHost->nP = nP;
                wuHost->iP = iP;
                wuHost->nI = nInteract > nIntPerTB ? nIntPerTB : nInteract;
                wuHost->iO = iO;
                iO += nP;
                nInteract -= wuHost->nI;
                ++wuHost;
                ++iI;
                nBufferOut += nP * sizeof(ppResult);
                }
            assert(nInteract==0);

            // Copy in nP particles
            for(j=0; j<nP; ++j) {
                partHost[j].dx =  pInfoIn[j].r[0];
                partHost[j].dy =  pInfoIn[j].r[1];
                partHost[j].dz =  pInfoIn[j].r[2];
                partHost[j].ax =  pInfoIn[j].a[0];
                partHost[j].ay =  pInfoIn[j].a[1];
                partHost[j].az =  pInfoIn[j].a[2];
                partHost[j].fSoft2 = pInfoIn[j].fSmooth2;
                /*partHost[j].dImaga = 0;*/
                }
            partHost += nPaligned;
            iP += nPaligned;
            }
        assert(iI == work->ppnBlocks/nBlkPer);
        ppResult *pCudaBufOut = reinterpret_cast<ppResult *>(work->pCudaBufOut);
        int nBufferIn = reinterpret_cast<char *>(partHost) - reinterpret_cast<char *>(work->pHostBuf);
        CUDA_CHECK(hipMemcpyAsync,(blkCuda, blkHost, nBufferIn, hipMemcpyHostToDevice, work->stream));
        dim3 dimBlock( WIDTH, 128/WIDTH );
        dim3 dimGrid( iI, 1,1);
        cudaInteract<nIntPerTB,WARPS,SYNC_RATE><<<dimGrid, dimBlock, 0, work->stream>>>(wuCuda,partCuda,blkCuda,pCudaBufOut );

        CUDA_CHECK(hipMemcpyAsync,(blkHost, work->pCudaBufOut, nBufferOut, hipMemcpyDeviceToHost, work->stream)
);
        CUDA_CHECK(hipEventRecord,(work->event,work->stream));

        work->next = cuda->wqCuda;
        cuda->wqCuda = work;

        *head = NULL;
        }
    }

extern "C"
void CUDA_sendWork(void *cudaCtx) {
    CUDACTX cuda = reinterpret_cast<CUDACTX>(cudaCtx);
    CUDA_sendWork< 128, ilpBlk<WIDTH> >(cuda,&cuda->nodePP);
    CUDA_sendWork< 128, ilcBlk<WIDTH> >(cuda,&cuda->nodePC);
    }


/*
** The following routines copy interactions from an ILP or ILC
** to a CUDA interaction block. The sizes may be different.
*/
template<int n>
int copyBLKs(ilpBlk<n> *out, ILP_BLK *in,int nIlp) {
    int i;
    if (n==ILP_PART_PER_BLK) {
        int nBlk = (nIlp+n-1) / n;
        for(i=0; i<nBlk; ++i) {
            memcpy(&out[i],&in[i],sizeof(out[i]));
            }
        return nBlk;
        }
    else {
        for(i=0; i<nIlp; ++i) {
            out[i/n].dx[i%n] = in[i/ILP_PART_PER_BLK].dx.f[i%ILP_PART_PER_BLK];
            out[i/n].dy[i%n] = in[i/ILP_PART_PER_BLK].dy.f[i%ILP_PART_PER_BLK];
            out[i/n].dz[i%n] = in[i/ILP_PART_PER_BLK].dz.f[i%ILP_PART_PER_BLK];
            out[i/n].m[i%n] = in[i/ILP_PART_PER_BLK].m.f[i%ILP_PART_PER_BLK];
            out[i/n].fourh2[i%n] = in[i/ILP_PART_PER_BLK].fourh2.f[i%ILP_PART_PER_BLK];
            }
        }
    return 0;
    }

template<int n>
int copyBLKs(ilcBlk<n> *out, ILC_BLK *in,int nIlp) {
    int i;
    if (n==ILP_PART_PER_BLK) {
        int nBlk = (nIlp+n-1) / n;
        for(i=0; i<nBlk; ++i) {
            memcpy(&out[i],&in[i],sizeof(out[i]));
            }
        return nBlk;
        }
    else {
        for(i=0; i<nIlp; ++i) {
            out[i/n].dx[i%n] = in[i/ILC_PART_PER_BLK].dx.f[i%ILC_PART_PER_BLK];
            out[i/n].dy[i%n] = in[i/ILC_PART_PER_BLK].dy.f[i%ILC_PART_PER_BLK];
            out[i/n].dz[i%n] = in[i/ILC_PART_PER_BLK].dz.f[i%ILC_PART_PER_BLK];
            out[i/n].xxxx[i%n] = in[i/ILC_PART_PER_BLK].xxxx.f[i%ILC_PART_PER_BLK];
            out[i/n].xxxy[i%n] = in[i/ILC_PART_PER_BLK].xxxy.f[i%ILC_PART_PER_BLK];
            out[i/n].xxxz[i%n] = in[i/ILC_PART_PER_BLK].xxxz.f[i%ILC_PART_PER_BLK];
            out[i/n].xxyz[i%n] = in[i/ILC_PART_PER_BLK].xxyz.f[i%ILC_PART_PER_BLK];
            out[i/n].xxyy[i%n] = in[i/ILC_PART_PER_BLK].xxyy.f[i%ILC_PART_PER_BLK];
            out[i/n].yyyz[i%n] = in[i/ILC_PART_PER_BLK].yyyz.f[i%ILC_PART_PER_BLK];
            out[i/n].xyyz[i%n] = in[i/ILC_PART_PER_BLK].xyyz.f[i%ILC_PART_PER_BLK];
            out[i/n].xyyy[i%n] = in[i/ILC_PART_PER_BLK].xyyy.f[i%ILC_PART_PER_BLK];
            out[i/n].yyyy[i%n] = in[i/ILC_PART_PER_BLK].yyyy.f[i%ILC_PART_PER_BLK];
        
            out[i/n].xxx[i%n] = in[i/ILC_PART_PER_BLK].xxx.f[i%ILC_PART_PER_BLK];
            out[i/n].xyy[i%n] = in[i/ILC_PART_PER_BLK].xyy.f[i%ILC_PART_PER_BLK];
            out[i/n].xxy[i%n] = in[i/ILC_PART_PER_BLK].xxy.f[i%ILC_PART_PER_BLK];
            out[i/n].yyy[i%n] = in[i/ILC_PART_PER_BLK].yyy.f[i%ILC_PART_PER_BLK];
            out[i/n].xxz[i%n] = in[i/ILC_PART_PER_BLK].xxz.f[i%ILC_PART_PER_BLK];
            out[i/n].yyz[i%n] = in[i/ILC_PART_PER_BLK].yyz.f[i%ILC_PART_PER_BLK];
            out[i/n].xyz[i%n] = in[i/ILC_PART_PER_BLK].xyz.f[i%ILC_PART_PER_BLK];
        
            out[i/n].xx[i%n] = in[i/ILC_PART_PER_BLK].xx.f[i%ILC_PART_PER_BLK];
            out[i/n].xy[i%n] = in[i/ILC_PART_PER_BLK].xy.f[i%ILC_PART_PER_BLK];
            out[i/n].xz[i%n] = in[i/ILC_PART_PER_BLK].xz.f[i%ILC_PART_PER_BLK];
            out[i/n].yy[i%n] = in[i/ILC_PART_PER_BLK].yy.f[i%ILC_PART_PER_BLK];
            out[i/n].yz[i%n] = in[i/ILC_PART_PER_BLK].yz.f[i%ILC_PART_PER_BLK];
        
            out[i/n].x[i%n] = in[i/ILC_PART_PER_BLK].x.f[i%ILC_PART_PER_BLK];
            out[i/n].y[i%n] = in[i/ILC_PART_PER_BLK].y.f[i%ILC_PART_PER_BLK];
            out[i/n].z[i%n] = in[i/ILC_PART_PER_BLK].z.f[i%ILC_PART_PER_BLK];
        
            out[i/n].m[i%n] = in[i/ILC_PART_PER_BLK].m.f[i%ILC_PART_PER_BLK];
            out[i/n].u[i%n] = in[i/ILC_PART_PER_BLK].u.f[i%ILC_PART_PER_BLK];
            }
        }
    return 0;
    }



// nIntPer: number of interactions handled per work unit: e.g., 128
template<int nIntPerTB, typename TILE,typename BLK>
int CUDA_queue(CUDACTX cuda,CUDAwqNode **head,workParticle *wp, TILE tile) {
    CUDAwqNode *work = *head;
    const int nBlkPer = nIntPerTB / WIDTH;
    const int nP = wp->nP;
    const int nInteract = tile->lstTile.nBlocks*ILP_PART_PER_BLK + tile->lstTile.nInLast;
    const int nPaligned = (nP+NP_ALIGN_MASK) & ~NP_ALIGN_MASK;
    const int nBlocks = tile->lstTile.nBlocks + (tile->lstTile.nInLast?1:0);
    const int nBlocksAligned = (nBlocks + nBlkPer - 1) & ~(nBlkPer - 1);
    const int nWork = nBlocksAligned / nBlkPer;
    const int nBytesIn = nPaligned * sizeof(ppInput) + nBlocksAligned*sizeof(BLK) + nWork*sizeof(ppWorkUnit);
    const int nBytesOut = nP * sizeof(ppResult) * nWork;

    assert(nWork*nBlkPer == nBlocksAligned);

    // Figure out the total amount of space we need, and see if there is enough
    // in the current block. If not, send it to the GPU and get another.
    // Space: nPaligned * 4 (float) * 7 (coordinates)
    //        nBlocks * sizeof(ILP_BLK)
    //
    if (work!=NULL && (work->ppSizeIn + nBytesIn + 8*sizeof(ppWorkUnit) > cuda->inCudaBufSize || work->ppSizeOut + nBytesOut > cuda->outCudaBufSize) ) {
        CUDA_sendWork<128,BLK>(cuda,head);
        work = NULL;
        assert(*head==NULL);
        }

    // If we don't have a PP work element, try to grab one
    if (work==NULL) {
        *head = work = getNode(cuda);
        if (work==NULL) return 0;
        work->checkFcn = CUDAcheckWorkInteraction<nIntPerTB>;
        work->ppSizeIn = 0;
        work->ppSizeOut = 0;
        work->ppnBuffered = 0;
        work->ppnBlocks = 0;
        }
    if (work==NULL) return 0;

    // Copy in the interactions. The ILP tiles can then be freed/reused.
    BLK *blk = reinterpret_cast<BLK *>(work->pHostBuf);
    //for(i=0; i<nBlocks; ++i) blk[work->ppnBlocks++] = tile->blk[i];
    copyBLKs(blk+work->ppnBlocks,tile->blk,nInteract);
    work->ppnBlocks += nBlocksAligned;

    work->ppSizeIn += nBytesIn;
    work->ppSizeOut += nBytesOut;
    work->ppNI[work->ppnBuffered] = nInteract;
    work->ppWP[work->ppnBuffered] = wp;
    ++wp->nRefs;

    if ( ++work->ppnBuffered == CUDA_PP_MAX_BUFFERED) CUDA_sendWork<nIntPerTB,BLK>(cuda,head);

    return 1;
    }

extern "C"
int CUDA_queuePP(void *cudaCtx,workParticle *wp, ILPTILE tile) {
    CUDACTX cuda = reinterpret_cast<CUDACTX>(cudaCtx);
    return CUDA_queue< 128, ILPTILE,ilpBlk<WIDTH> >(cuda,&cuda->nodePP,wp,tile);
    }

extern "C"
int CUDA_queuePC(void *cudaCtx,workParticle *wp, ILCTILE tile) {
    CUDACTX cuda = reinterpret_cast<CUDACTX>(cudaCtx);
    return CUDA_queue< 128, ILCTILE,ilcBlk<WIDTH> >(cuda,&cuda->nodePC,wp,tile);
    }

