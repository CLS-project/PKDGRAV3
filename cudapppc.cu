#include "hip/hip_runtime.h"
/*  This file is part of PKDGRAV3 (http://www.pkdgrav.org/).
 *  Copyright (c) 2001-2018 Joachim Stadel & Douglas Potter
 *
 *  PKDGRAV3 is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  PKDGRAV3 is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with PKDGRAV3.  If not, see <http://www.gnu.org/licenses/>.
 */

/* -*- Mode: C++; tab-width: 8; indent-tabs-mode: nil; c-basic-offset: 4 -*- */
#ifdef HAVE_CONFIG_H
#include "config.h"
#else
#include "pkd_config.h"
#endif
#include <stdio.h>
#include "cudapppc.h"
#include "pp.h"
#include "pc.h"

#define SYNC_RATE 16  // Must be: 1, 2, 4, 8, 16
#define WIDTH 32

#define TB_THREADS 128
#define WARPS (TB_THREADS/32)

#define PP_WU 128
#define PC_WU 32

#include "ilp.h"
#include "ilc.h"
#include "basetype.h"

/*
** The following are the basically the same as ILP_BLK and ILC_BLK,
** but we need to be able to alter their sizes.
*/

template<int n>
struct ilpBlk {
    float dx[n], dy[n], dz[n];    /* Offset from ilp->cx, cy, cz */
    float m[n];             /* Mass */
    float fourh2[n];        /* Softening: calculated */
    };

template<int n>
struct ilcBlk {
    float dx[n],dy[n],dz[n];
    float xxxx[n],xxxy[n],xxxz[n],xxyz[n],xxyy[n],yyyz[n],xyyz[n],xyyy[n],yyyy[n];
    float xxx[n],xyy[n],xxy[n],yyy[n],xxz[n],yyz[n],xyz[n];
    float xx[n],xy[n],xz[n],yy[n],yz[n];
    float x[n],y[n],z[n];
    float m[n],u[n];
    };

// One of these entries for each interaction block
struct ppWorkUnit {
    uint32_t nP;   // Number of particles
    uint32_t nI;   // Number of interactions in the block
    uint32_t iP;   // Index of first particle
//    uint32_t iB;   // Index of the interaction block
    uint32_t iO;   // Index of the output block
    };


struct ppInput {
    float dx, dy, dz;
    float ax, ay, az;
    float fSoft2;
    float dImaga;
    };

/* Each thread block outputs this for each particle */
struct __align__(32) ppResult {
    float ax;
    float ay;
    float az;
    float fPot;
    float dirsum;
    float normsum;
    };

#define NP_ALIGN (128/sizeof(ppResult))
#define NP_ALIGN_MASK (NP_ALIGN-1)

template<typename BLK>
static void dumpWork(struct cuda_wq_node *work) {
    BLK * __restrict__ blkHost = reinterpret_cast<BLK*>(work->pHostBufToGPU);
    ppWorkUnit * __restrict__ wuHost = reinterpret_cast<ppWorkUnit *>(blkHost + work->ppnBlocks);
    int i;
    fprintf(stderr,"kernel %s has %d work units\n", work->kernelName,work->pppc.nGrid);
    for( i=0; i<work->pppc.nGrid; ++i) {
        fprintf(stderr,"%4d %5d %5d %5d\n", i, wuHost[i].nP, wuHost[i].nI, wuHost[i].iP/*, wuHost[i].iB*/);
        }

    }

/*
** Occupancy (theoretical):
**   Compute 2.x:  8 (blocks) * PP_THREADS (128) = 1024 (2/3 MAX)
**   Compute 3.x: 16 (blocks) * PP_THREADS (128) = 2048 (MAX!)
**
** To reduce memory, we syncthreads() and flush the particles
** results every SYNC_RATE particles => 8 seems a good choice.
**1
** Shared memory requirements
**  - Particles         32 * nSyncRate (16)                =  512
**  - warp reduction     4 * nWarps (4) * 32 (threads)     =  512
**  - warp results      24 * nSyncRate (16)  * nWarps (4)  = 1536
** TOTAL 2560 * 16 blocks = 40 KB
**
** nvcc -DHAVE_CONFIG_H --ptxas-options=-v -c  -I. -arch=sm_20 cudapp.cu
** ptxas info    : 11 bytes gmem, 8 bytes cmem[14]
** ptxas info    : Compiling entry function '_Z6cudaPPILi4ELi16EEvPK10ppWorkUnitPK7ppInputPK7ILP_BLKP8ppResult' for 'sm_20'
** ptxas info    : Function properties for _Z6cudaPPILi4ELi16EEvPK10ppWorkUnitPK7ppInputPK7ILP_BLKP8ppResult
**     0 bytes stack frame, 0 bytes spill stores, 0 bytes spill loads
** ptxas info    : Used 35 registers, 2560 bytes smem, 64 bytes cmem[0], 12 bytes cmem[16]
**
** Interaction lists limits.
**   Compute 2.x: 65535 * PP_THREADS (128) = 8 million
**   Compute 3.x: effectively unlimited
**
*/

// A good number for nWarps is 4 giving 128 threads per thread block, nSyncRate=8
// Each thread block outputs ay,ay,az,fPot,dirsum,normsum for each particle
template <int nWarps,int nWarpsPerWU,int nSyncRate,int bGravStep>
__global__ void cudaInteract(
    const ppWorkUnit * __restrict__ work,
    const ppInput * __restrict__ pPart,
    const ilpBlk<WIDTH> * __restrict__ blk,
    ppResult *out) {
    int i, iSync;
    int iWork, iI, iWarp;

    if (nWarpsPerWU==1) {           // blockDim.z == nWarps, blockDim.y == 1, blockDim.x == 32
        iWork = blockIdx.x * nWarps + threadIdx.z; // Work and corresponding blk
        iI = threadIdx.x; // Index into blk
        iWarp = threadIdx.z;
        }
    else if (nWarps==nWarpsPerWU) { // blockDim.z == 1, blockDim.y == nWarps, blockDim.x == 32
        iWork = blockIdx.x; // Index of work and blk
        iI =   threadIdx.y*blockDim.x + threadIdx.x; // Index of interaction
        iWarp = threadIdx.y;
        }
    else {                          // blockDim.z == 2, blockDim.y == 2, blockDim.x == 32
        // Calculate our interaction and particle group
        iWork = blockIdx.x*blockDim.z + threadIdx.z; // Work and corresponding blk
        iI =   threadIdx.y*blockDim.x + threadIdx.x; // Thread working on blk
        iWarp = threadIdx.y + blockDim.y*threadIdx.z;
        }
    int iTinW = iI % 32;

    uint32_t nP = work[iWork].nP; // Number of particles
    pPart += work[iWork].iP; // First particle
    uint32_t nI = work[iWork].nI; // Number of interactions
//    blk += work[iWork].iB*blockDim.y + threadIdx.y; // blk[threadIdx.x] is our interaction
    blk += iWork*blockDim.y + threadIdx.y; // blk[threadIdx.x] is our interaction
    out += work[iWork].iO;   // Result for each particle

    __shared__ union {
        ppInput P[nWarps/nWarpsPerWU][nSyncRate];
        float   W[nWarps/nWarpsPerWU][nSyncRate*sizeof(ppInput)/sizeof(float)];
        } Particles;

    __shared__ float wX[nSyncRate][nWarps];
    __shared__ float wY[nSyncRate][nWarps];
    __shared__ float wZ[nSyncRate][nWarps];

    __shared__ float wPot[nSyncRate][nWarps];
    __shared__ float wDirsum[nSyncRate][nWarps];
    __shared__ float wNormsum[nSyncRate][nWarps];


    // Load the interaction. It is blocked for performance.
    float iX,iY,iZ,iM,ifourh2;
    if (iI < nI) {
        iX = blk->dx[threadIdx.x];
        iY = blk->dy[threadIdx.x];
        iZ = blk->dz[threadIdx.x];
        iM = blk->m[threadIdx.x];
        ifourh2 = blk->fourh2[threadIdx.x];
        }

    // Apply the particles, nSyncRate at a time
    for(iSync=0; iSync<nP; iSync += nSyncRate) {
        int iEnd = nP - iSync;
        if (iEnd > nSyncRate) iEnd=nSyncRate;
        // Preload the bucket of particles - this is a memcpy
        if (iI < iEnd*sizeof(ppInput) / sizeof(float)) {
            Particles.W[threadIdx.z][iI] = (reinterpret_cast<const float *>(pPart+iSync))[iI];
            }
        if (nWarpsPerWU>1) __syncthreads();
        if (iI < iEnd && bGravStep) { 
            float ax = Particles.P[threadIdx.z][iI].ax;
            float ay = Particles.P[threadIdx.z][iI].ay;
            float az = Particles.P[threadIdx.z][iI].az;
            ax = ax*ax + ay*ay + az*az;
            if (ax > 0.0f) ax = rsqrtf(ax);
            Particles.P[threadIdx.z][iI].dImaga = ax;
            }
        if (nWarpsPerWU>1) __syncthreads();

        for( i=0; i<iEnd; ++i) {
            float ax=0.0f, ay=0.0f, az=0.0f, fPot=0.0f, dir=0.0f, norm=0.0f;
            if (iI < nI) {
                float Px = Particles.P[threadIdx.z][i].dx;
                float Py = Particles.P[threadIdx.z][i].dy;
                float Pz = Particles.P[threadIdx.z][i].dz;
		float fSoft2 = Particles.P[threadIdx.z][i].fSoft2;
		float iax = Particles.P[threadIdx.z][i].ax;
		float iay = Particles.P[threadIdx.z][i].ay;
		float iaz = Particles.P[threadIdx.z][i].az;
		float imaga = Particles.P[threadIdx.z][i].dImaga;
		EvalPP<float,bool,true>(
		    Px, Py, Pz, fSoft2, 
		    iX, iY, iZ, ifourh2, iM,
		    ax, ay, az, fPot,
		    iax, iay, iaz, imaga,
		    dir, norm);
                }
            // Horizontal add within each warp -- no sychronization required
            warpReduceAndStore<float,32>(iTinW,ax,       &wX[i][iWarp]);
            warpReduceAndStore<float,32>(iTinW,ay,       &wY[i][iWarp]);
            warpReduceAndStore<float,32>(iTinW,az,       &wZ[i][iWarp]);
            warpReduceAndStore<float,32>(iTinW,fPot,     &wPot[i][iWarp]);
            if (bGravStep) {
                warpReduceAndStore<float,32>(iTinW,dir,  &wDirsum[i][iWarp]);
                warpReduceAndStore<float,32>(iTinW,norm, &wNormsum[i][iWarp]);
                }
            }

        if (nWarpsPerWU>1) __syncthreads();
        // Assuming four warps & SYNC of 8, the cache looks like this:
        //                0     1     2     3       4
        // Cache: 1x4x8   P0    P0    P0    P0      P1 P1 P1 P1 P2 ... P6 P7 P7 P7 P7
        // Cache: 4x1x8   P0,I0 P0,I1 P0,I2 P0,I3   P1,I0 ... 
        // Cache: 2x2x8   P0,I0 P0,I0 P0,I1 P0,I1   P1,I0
        // every set of 4 threads does another reduce. As long as the
        // number of warps is a power of 2 <= the warp size (32), we
        // can do this step without any further synchronization.
        int nOut = iEnd * nWarpsPerWU; // Normally 64
        if (iI<nOut) {
            int iP    = (iI & ~(nWarpsPerWU-1)) * nWarps/nWarpsPerWU + threadIdx.z*nWarpsPerWU; // 0,4,8,...
            int iWarp = iI &  (nWarpsPerWU-1); // 0 .. 3
            int iOut  = iI / nWarpsPerWU + iSync;
         
            warpReduceAndStore<float,nWarpsPerWU>(      &wX[0][0]+iP,iWarp,&out[iOut].ax);
            warpReduceAndStore<float,nWarpsPerWU>(      &wY[0][0]+iP,iWarp,&out[iOut].ay);
            warpReduceAndStore<float,nWarpsPerWU>(      &wZ[0][0]+iP,iWarp,&out[iOut].az);
            warpReduceAndStore<float,nWarpsPerWU>(    &wPot[0][0]+iP,iWarp,&out[iOut].fPot);
            if (bGravStep) {
                warpReduceAndStore<float,nWarpsPerWU>( &wDirsum[0][0]+iP,iWarp,&out[iOut].dirsum);
                warpReduceAndStore<float,nWarpsPerWU>(&wNormsum[0][0]+iP,iWarp,&out[iOut].normsum);
                }
            }
        }
    }

template <int nWarps,int nWarpsPerWU,int nSyncRate,int bGravStep>
__global__ void cudaInteract(
    const ppWorkUnit * __restrict__ work,
    const ppInput * __restrict__ pPart,
    const ilcBlk<WIDTH> * __restrict__ blk,
    ppResult *out) {
    int i, iSync;
    int iWork, iI, iWarp;

    if (nWarpsPerWU==1) {           // blockDim.z == nWarps, blockDim.y == 1, blockDim.x == 32
        iWork = blockIdx.x * nWarps + threadIdx.z; // Work and corresponding blk
        iI = threadIdx.x; // Index into blk
        iWarp = threadIdx.z;
        }
    else if (nWarps==nWarpsPerWU) { // blockDim.z == 1, blockDim.y == nWarps, blockDim.x == 32
        iWork = blockIdx.x; // Index of work and blk
        iI =   threadIdx.y*blockDim.x + threadIdx.x; // Index of interaction
        iWarp = threadIdx.y;
        }
    else {
        // Calculate our interaction and particle group
        iWork = blockIdx.x*blockDim.z + threadIdx.z; // Work and corresponding blk
        iI =   threadIdx.y*blockDim.x + threadIdx.x; // Thread working on blk
        int iAll = iI + threadIdx.z*blockDim.y*blockDim.x;
        iWarp = iAll / 32;
        }
    int iTinW = iI % 32;

    int nP = work[iWork].nP; // Number of particles
    pPart += work[iWork].iP; // First particle
    int nI = work[iWork].nI; // Number of interactions
//    blk += work[iWork].iB*blockDim.y + threadIdx.y; // blk[threadIdx.x] is our interaction
    blk += iWork*blockDim.y + threadIdx.y; // blk[threadIdx.x] is our interaction
    out += work[iWork].iO;   // Result for each particle

    __shared__ union {
        ppInput P[nWarps/nWarpsPerWU][nSyncRate];
        float   W[nWarps/nWarpsPerWU][nSyncRate*sizeof(ppInput)/sizeof(float)];
        } Particles;

    __shared__ float wX[nSyncRate][nWarps];
    __shared__ float wY[nSyncRate][nWarps];
    __shared__ float wZ[nSyncRate][nWarps];

    __shared__ float wPot[nSyncRate][nWarps];
    __shared__ float wDirsum[nSyncRate][nWarps];
    __shared__ float wNormsum[nSyncRate][nWarps];


    // Load the interaction. It is blocked for performance.
    float Idx,Idy,Idz;
    float Ixxxx,Ixxxy,Ixxxz,Ixxyz,Ixxyy,Iyyyz,Ixyyz,Ixyyy,Iyyyy;
    float Ixxx,Ixyy,Ixxy,Iyyy,Ixxz,Iyyz,Ixyz;
    float Ixx,Ixy,Ixz,Iyy,Iyz;
#ifdef USE_DIAPOLE
    float Ix,Iy,Iz;
#endif
    float Im,Iu;
    if (iI < nI) {
        Idx = blk->dx[threadIdx.x];
        Idy = blk->dy[threadIdx.x];
        Idz = blk->dz[threadIdx.x];
        Ixxxx = blk->xxxx[threadIdx.x];
        Ixxxy = blk->xxxy[threadIdx.x];
        Ixxxz = blk->xxxz[threadIdx.x];
        Ixxyz = blk->xxyz[threadIdx.x];
        Ixxyy = blk->xxyy[threadIdx.x];
        Iyyyz = blk->yyyz[threadIdx.x];
        Ixyyz = blk->xyyz[threadIdx.x];
        Ixyyy = blk->xyyy[threadIdx.x];
        Iyyyy = blk->yyyy[threadIdx.x];
        Ixxx = blk->xxx[threadIdx.x];
        Ixyy = blk->xyy[threadIdx.x];
        Ixxy = blk->xxy[threadIdx.x];
        Iyyy = blk->yyy[threadIdx.x];
        Ixxz = blk->xxz[threadIdx.x];
        Iyyz = blk->yyz[threadIdx.x];
        Ixyz = blk->xyz[threadIdx.x];
        Ixx = blk->xx[threadIdx.x];
        Ixy = blk->xy[threadIdx.x];
        Ixz = blk->xz[threadIdx.x];
        Iyy = blk->yy[threadIdx.x];
        Iyz = blk->yz[threadIdx.x];
#ifdef USE_DIAPOLE
        Ix = blk->x[threadIdx.x];
        Iy = blk->y[threadIdx.x];
        Iz = blk->z[threadIdx.x];
#endif
        Im = blk->m[threadIdx.x];
        Iu = blk->u[threadIdx.x];
        }
    for(iSync=0; iSync<nP; iSync += nSyncRate) {
        int iEnd = nP - iSync;
        if (iEnd > nSyncRate) iEnd=nSyncRate;
        // Preload the bucket of particles - this is a memcpy
        if (iI < iEnd*sizeof(ppInput) / sizeof(float)) {
            Particles.W[threadIdx.z][iI] = (reinterpret_cast<const float *>(pPart+iSync))[iI];
            }
        if (nWarpsPerWU>1) __syncthreads();
        if (iI < iEnd && bGravStep) { 
            float ax = Particles.P[threadIdx.z][iI].ax;
            float ay = Particles.P[threadIdx.z][iI].ay;
            float az = Particles.P[threadIdx.z][iI].az;
            Particles.P[threadIdx.z][iI].dImaga = ax*ax + ay*ay + az*az;
            if (Particles.P[threadIdx.z][iI].dImaga > 0.0f)
                Particles.P[threadIdx.z][iI].dImaga = rsqrtf(Particles.P[threadIdx.z][iI].dImaga);
            }
        if (nWarpsPerWU>1) __syncthreads();

        for( i=0; i<iEnd; ++i) {
            float ax=0.0f, ay=0.0f, az=0.0f, fPot=0.0f, dirsum=0.0f, norm=0.0f;
            if (iI < nI) {
                float Pdx = Particles.P[threadIdx.z][i].dx;
                float Pdy = Particles.P[threadIdx.z][i].dy;
                float Pdz = Particles.P[threadIdx.z][i].dz;
		float Pax = Particles.P[threadIdx.z][i].ax;
		float Pay = Particles.P[threadIdx.z][i].ay;
		float Paz = Particles.P[threadIdx.z][i].az;
		float fSoft2 = Particles.P[threadIdx.z][i].fSoft2;
		float Pimaga = Particles.P[threadIdx.z][i].dImaga;
		EvalPC<float,bool,true>(
		    Pdx, Pdy, Pdz,fSoft2,
		    Idx, Idy, Idz, Im, Iu,
		    Ixxxx, Ixxxy, Ixxxz, Ixxyz, Ixxyy, Iyyyz, Ixyyz, Ixyyy, Iyyyy,
		    Ixxx, Ixyy, Ixxy, Iyyy, Ixxz, Iyyz, Ixyz, Ixx, Ixy, Ixz, Iyy, Iyz,
#ifdef USE_DIAPOLE
		    Ix, Iy, Iz,
#endif
		    ax, ay, az, fPot,
		    Pax, Pay, Paz, Pimaga,
		    dirsum, norm);
                }
            // Horizontal add within each warp -- no sychronization required
            warpReduceAndStore<float,32>(iTinW,ax,     &wX[i][iWarp]);
            warpReduceAndStore<float,32>(iTinW,ay,     &wY[i][iWarp]);
            warpReduceAndStore<float,32>(iTinW,az,     &wZ[i][iWarp]);
            warpReduceAndStore<float,32>(iTinW,fPot,   &wPot[i][iWarp]);
            if (bGravStep) {
                warpReduceAndStore<float,32>(iTinW,dirsum, &wDirsum[i][iWarp]);
                warpReduceAndStore<float,32>(iTinW,norm,&wNormsum[i][iWarp]);
                }
	}

        if (nWarpsPerWU>1) __syncthreads();
        // Assuming four warps & SYNC of 8, the cache looks like this:
        //                0     1     2     3       4
        // Cache: 1x4x8   P0    P0    P0    P0      P1 P1 P1 P1 P2 ... P6 P7 P7 P7 P7
        // Cache: 4x1x8   P0,I0 P0,I1 P0,I2 P0,I3   P1,I0 ... 
        // Cache: 2x2x8   P0,I0 P0,I0 P0,I1 P0,I1   P1,I0
        // every set of 4 threads does another reduce. As long as the
        // number of warps is a power of 2 <= the warp size (32), we
        // can do this step without any further synchronization.
        int nOut = iEnd * nWarpsPerWU; // Normally 64
        if (iI<nOut) {
            int iP    = (iI & ~(nWarpsPerWU-1)) * nWarps/nWarpsPerWU + threadIdx.z; // 0,4,8,...
            int iWarp = iI &  (nWarpsPerWU-1); // 0 .. 3
            int iOut  = iI / nWarpsPerWU + iSync;
         
            warpReduceAndStore<float,nWarpsPerWU>(      &wX[0][iP],iWarp,&out[iOut].ax);
            warpReduceAndStore<float,nWarpsPerWU>(      &wY[0][iP],iWarp,&out[iOut].ay);
            warpReduceAndStore<float,nWarpsPerWU>(      &wZ[0][iP],iWarp,&out[iOut].az);
            warpReduceAndStore<float,nWarpsPerWU>(    &wPot[0][iP],iWarp,&out[iOut].fPot);
            if (bGravStep) {
                warpReduceAndStore<float,nWarpsPerWU>( &wDirsum[0][iP],iWarp,&out[iOut].dirsum);
                warpReduceAndStore<float,nWarpsPerWU>(&wNormsum[0][iP],iWarp,&out[iOut].normsum);
                }
            }
        }
    }

extern "C"
void pkdParticleWorkDone(workParticle *wp);

template<int nIntPerWU>
int CUDAcheckWorkInteraction( void *vpp, void *vwork ) {
    CUDAwqNode *work = reinterpret_cast<CUDAwqNode *>(vwork);
    ppResult *pR       = reinterpret_cast<ppResult *>(work->pHostBufFromGPU);
    int ib, iw, ip;

    for( ib=0; ib<work->ppnBuffered; ++ib) {
        workParticle *wp = work->ppWP[ib];
        PINFOOUT *pInfoOut = wp->pInfoOut;
        int nWork = (work->ppNI[ib] + nIntPerWU - 1) / nIntPerWU;
        for(iw=0; iw<nWork; ++iw) {
            for(ip=0; ip<wp->nP; ++ip) {
                pInfoOut[ip].a[0]    += pR[ip].ax;
                pInfoOut[ip].a[1]    += pR[ip].ay;
                pInfoOut[ip].a[2]    += pR[ip].az;
                pInfoOut[ip].fPot    += pR[ip].fPot;
                pInfoOut[ip].dirsum  += pR[ip].dirsum;
                pInfoOut[ip].normsum += pR[ip].normsum;
                }
            pR += wp->nP;
            }
        pkdParticleWorkDone(wp);

        }
    return 0;
    }

extern "C"
void CUDAsetupPP(void) {
    //hipFuncSetCacheConfig(reinterpret_cast<const void*>(cudaPP),hipFuncCachePreferL1);
    }

/* If this returns an error, then the caller must attempt recovery or abort */
template<int nIntPerTB, int nIntPerWU, typename BLK>
int initWork( void *ve, void *vwork ) {
    CUDAwqNode *work = reinterpret_cast<CUDAwqNode *>(vwork);
    const int nBlkPer = nIntPerWU / WIDTH;
    const int nWork = work->ppnBlocks/nBlkPer;

    // The interation blocks -- already copied to the host memory
    BLK * __restrict__ blkCuda = reinterpret_cast<BLK*>(work->pCudaBufIn);

    // The interaction block descriptors
    ppWorkUnit * __restrict__ wuCuda = reinterpret_cast<ppWorkUnit *>(blkCuda + work->ppnBlocks);

    // The particle information
    ppInput * __restrict__ partCuda = reinterpret_cast<ppInput *>(wuCuda + ((nWork+7)&~7));

    ppResult *pCudaBufOut = reinterpret_cast<ppResult *>(work->pCudaBufOut);

    CUDA_RETURN(hipMemcpyAsync,(blkCuda, work->pHostBufToGPU, work->pppc.nBufferIn, hipMemcpyHostToDevice, work->stream));
    CUDA_RETURN(hipEventRecord,(work->eventCopyDone,work->stream));

    dim3 dimBlock( WIDTH, nIntPerWU/WIDTH, nIntPerTB/nIntPerWU );
    dim3 dimGrid( work->pppc.nGrid, 1,1);
    work->dimBlock = dimBlock;
    work->dimGrid = dimGrid;
    if (work->bGravStep) {
        cudaInteract<WARPS,nIntPerWU/32,SYNC_RATE*nIntPerWU/nIntPerTB,1>
            <<<dimGrid, dimBlock, 0, work->stream>>>
            (wuCuda,partCuda,blkCuda,pCudaBufOut );
        }
    else {
        cudaInteract<WARPS,nIntPerWU/32,SYNC_RATE*nIntPerWU/nIntPerTB,0>
            <<<dimGrid, dimBlock, 0, work->stream>>>
            (wuCuda,partCuda,blkCuda,pCudaBufOut );
        }
    CUDA_RETURN(hipEventRecord,(work->eventKernelDone,work->stream));
    CUDA_RETURN(hipMemcpyAsync,(work->pHostBufFromGPU, work->pCudaBufOut, work->pppc.nBufferOut, hipMemcpyDeviceToHost, work->stream) );
#ifdef USE_CUDA_EVENTS
    CUDA_RETURN(hipEventRecord,(work->event,work->stream));
#endif

    return hipSuccess;
    }

template<int nIntPerTB, int nIntPerWU, typename BLK>
void CUDA_sendWork(CUDACTX cuda,CUDAwqNode **head) {
    CUDAwqNode *work = *head;
    if (work != NULL) {
        int i, j;
        int iI=0, iP=0, iO=0;
        const int nBlkPer = nIntPerWU / WIDTH;
        const int nWork = work->ppnBlocks/nBlkPer;

        // The interation blocks -- already copied to the host memory
        BLK * __restrict__ blkHost = reinterpret_cast<BLK*>(work->pHostBufToGPU);

        // The interaction block descriptors
        ppWorkUnit * __restrict__ wuHost = reinterpret_cast<ppWorkUnit *>(blkHost + work->ppnBlocks);

        // The particle information
        ppInput * __restrict__ partHost = reinterpret_cast<ppInput *>(wuHost + ((nWork+7)&~7));

        work->pppc.nBufferOut = 0;
        for( i=0; i<work->ppnBuffered; ++i) {
            const int nP = work->ppWP[i]->nP;
            PINFOIN *pInfoIn = work->ppWP[i]->pInfoIn;
            int nPaligned = (nP+NP_ALIGN_MASK) & ~NP_ALIGN_MASK;
            int nInteract = work->ppNI[i];
            int nBlocks = (nInteract+nIntPerWU-1) / nIntPerWU;

            // Generate a interaction block descriptor for each block
            for(j=0; j<nBlocks; ++j) {
                wuHost->nP = nP;
                wuHost->iP = iP;
                wuHost->nI = nInteract > nIntPerWU ? nIntPerWU : nInteract;
                wuHost->iO = iO;
//                wuHost->iB = iI;
                iO += nP;
                nInteract -= wuHost->nI;
                ++wuHost;
                ++iI;
                work->pppc.nBufferOut += nP * sizeof(ppResult);
                }
            assert(nInteract==0);

            // Copy in nP particles
            for(j=0; j<nP; ++j) {
                partHost[j].dx =  pInfoIn[j].r[0];
                partHost[j].dy =  pInfoIn[j].r[1];
                partHost[j].dz =  pInfoIn[j].r[2];
                partHost[j].ax =  pInfoIn[j].a[0];
                partHost[j].ay =  pInfoIn[j].a[1];
                partHost[j].az =  pInfoIn[j].a[2];
                partHost[j].fSoft2 = pInfoIn[j].fSmooth2;
                /*partHost[j].dImaga = 0;*/
                }
            partHost += nPaligned;
            iP += nPaligned;
            }
        assert(iI == work->ppnBlocks/nBlkPer);
        /* Pad the work out so all work units have valid data */
        const int nWUPerTB = nIntPerTB/nIntPerWU;
        while((iI&(nWUPerTB-1)) != 0) {
            wuHost->nP = 0;
            wuHost->iP = 0;
            wuHost->nI = 0;
            wuHost->iO = iO;
            ++wuHost;
            ++iI;
            }

        assert((iI & (nWUPerTB-1)) == 0);
        work->pppc.nGrid = iI/nWUPerTB;
        work->pppc.nBufferIn = reinterpret_cast<char *>(partHost) - reinterpret_cast<char *>(work->pHostBufToGPU);

        work->startTime = CUDA_getTime();
#ifdef CUDA_STREAMS
        OPA_Queue_enqueue(cuda->queueWORK, work, CUDAwqNode, q.hdr);
#else
        work->q.next = cuda->wqCudaBusy;
        cuda->wqCudaBusy = work;
        hipError_t rc = static_cast<hipError_t>((*work->initFcn)(work->ctx,work));
        if ( rc != hipSuccess) CUDA_attempt_recovery(cuda,rc);
#endif
        *head = NULL;
        }
    }

extern "C"
void CUDA_sendWork(void *cudaCtx) {
    CUDACTX cuda = reinterpret_cast<CUDACTX>(cudaCtx);
    CUDA_sendWork< TB_THREADS, PP_WU, ilpBlk<WIDTH> >(cuda,&cuda->nodePP);
    CUDA_sendWork< TB_THREADS, PC_WU, ilcBlk<WIDTH> >(cuda,&cuda->nodePC);
    }

/*
** These functions allow us to send part (or all) of the work directly to the CPU.
** The idea is that "extra" bits get done directly instead of by the GPU.
*/
extern "C"
void pkdGravEvalPP(PINFOIN *pPart, int nBlocks, int nInLast, ILP_BLK *blk,  PINFOOUT *pOut );
extern "C"
void pkdGravEvalPC(PINFOIN *pPart, int nBlocks, int nInLast, ILC_BLK *blk,  PINFOOUT *pOut );

static void GravEval(PINFOIN *pPart, int nBlocks, int nInLast, ILP_BLK *blk,  PINFOOUT *pOut) {
    pkdGravEvalPP(pPart,nBlocks,nInLast,blk,pOut);
    }
static void GravEval(PINFOIN *pPart, int nBlocks, int nInLast, ILC_BLK *blk,  PINFOOUT *pOut) {
    pkdGravEvalPC(pPart,nBlocks,nInLast,blk,pOut);
    }

template<typename BLK>
static void finishGravity(workParticle *wp, int nBlocks, int nInLast, BLK *blk ) {
    int i;
    for(i=0; i<wp->nP; ++i ) {
        PINFOIN *pPart = &wp->pInfoIn[i];
        PINFOOUT *pOut = &wp->pInfoOut[i];
        GravEval(pPart,nBlocks,nInLast,blk,pOut);
        }
    ++wp->nRefs;
    pkdParticleWorkDone(wp);
    }

/*
** The following routines copy interactions from an ILP or ILC
** to a CUDA interaction block. The sizes may not be different.
*/
template<int n>
int copyBLKs(ilpBlk<n> *out, ILP_BLK *in,int nIlp) {
    assert(n==ILP_PART_PER_BLK);
    int i, nBlk = (nIlp+n-1) / n;
    for(i=0; i<nBlk; ++i) memcpy(&out[i],&in[i],sizeof(out[i]));
    return nBlk;
    }

template<int n>
int copyBLKs(ilcBlk<n> *out, ILC_BLK *in,int nIlp) {
    assert(n==ILC_PART_PER_BLK);
    int i, nBlk = (nIlp+n-1) / n;
    for(i=0; i<nBlk; ++i) memcpy(&out[i],&in[i],sizeof(out[i]));
    return nBlk;
    }

// nIntPer: number of interactions handled per work unit: e.g., 128
template<int nIntPerTB, int nIntPerWU, typename TILE,typename BLK>
int CUDA_queue(CUDACTX cuda,CUDAwqNode **head,workParticle *wp, TILE tile, int bGravStep, const char *kernelName) {
    /* Refuse the work if it looks like we will overwhelm the GPU with nonsense */
    if (cuda->nWorkQueueSize == 0) return 0;
    assert(cuda->nWorkQueueBusy >=0 && cuda->nWorkQueueBusy <= cuda->nWorkQueueSize);

    // If the work queue is half used, and there are fewer than 2 particles let the CPU handle it
//    if (cuda->nWorkQueueBusy > cuda->nWorkQueueSize-3 && wp->nP <= 2) return 0;

    CUDAwqNode *work = *head;
    /*const int nBlkPerTB = nIntPerTB / WIDTH;*/
    const int nBlkPerWU = nIntPerWU / WIDTH;
    const int nP = wp->nP;
    const int nPaligned = (nP+NP_ALIGN_MASK) & ~NP_ALIGN_MASK;
    const int nBlocks = tile->lstTile.nBlocks + (tile->lstTile.nInLast?1:0);
    int nBlocksAligned,nInteract;

    // If there are too few free blocks then the CPU does the "hair"
//    if (cuda->nWorkQueueBusy > cuda->nWorkQueueSize-3) {
//        nBlocksAligned = (tile->lstTile.nBlocks + (tile->lstTile.nInLast==32?1:0) ) & ~(nBlkPerWU - 1);
//        nInteract = nBlocksAligned * ILP_PART_PER_BLK;
//        }
//    else {
        nBlocksAligned = (nBlocks + nBlkPerWU - 1) & ~(nBlkPerWU - 1);
        nInteract = tile->lstTile.nBlocks*ILP_PART_PER_BLK + tile->lstTile.nInLast;
//        }
    const int nWork = nBlocksAligned / nBlkPerWU;
    const int nBytesIn = nPaligned * sizeof(ppInput) + nBlocksAligned*sizeof(BLK) + nWork*sizeof(ppWorkUnit);
    const int nBytesOut = nP * sizeof(ppResult) * nWork;

    assert(nWork*nBlkPerWU == nBlocksAligned);

    // Figure out the total amount of space we need, and see if there is enough
    // in the current block. If not, send it to the GPU and get another.
    // Space: nPaligned * 4 (float) * 7 (coordinates)
    //        nBlocks * sizeof(ILP_BLK)
    //
    if (work!=NULL && (work->ppSizeIn + nBytesIn + 8*sizeof(ppWorkUnit) > cuda->inCudaBufSize || work->ppSizeOut + nBytesOut > cuda->outCudaBufSize) ) {
        CUDA_sendWork<nIntPerTB,nIntPerWU,BLK>(cuda,head);
        work = NULL;
        assert(*head==NULL);
        }

    // If we don't have a PP work element, try to grab one
    if (work==NULL) {
        *head = work = getNode(cuda);
        if (work==NULL) return 0;
        work->ctx = NULL;
        work->doneFcn = CUDAcheckWorkInteraction<nIntPerWU>;
        work->initFcn = initWork<nIntPerTB,nIntPerWU,BLK>;
        work->dumpFcn = dumpWork<BLK>;
        work->kernelName = kernelName;
        work->ppSizeIn = 0;
        work->ppSizeOut = 0;
        work->ppnBuffered = 0;
        work->ppnBlocks = 0;
        work->bGravStep = bGravStep;
        }
    if (work==NULL) return 0;

    if (nBlocksAligned>0) {
        // Copy in the interactions. The ILP tiles can then be freed/reused.
        BLK *blk = reinterpret_cast<BLK *>(work->pHostBufToGPU);
        //for(i=0; i<nBlocks; ++i) blk[work->ppnBlocks++] = tile->blk[i];
        copyBLKs(blk+work->ppnBlocks,tile->blk,nInteract);
        work->ppnBlocks += nBlocksAligned;

        work->ppSizeIn += nBytesIn;
        work->ppSizeOut += nBytesOut;
        work->ppNI[work->ppnBuffered] = nInteract;
        work->ppWP[work->ppnBuffered] = wp;
        ++wp->nRefs;

        if ( ++work->ppnBuffered == CUDA_WP_MAX_BUFFERED) CUDA_sendWork<nIntPerTB,nIntPerWU,BLK>(cuda,head);
        }
    // Evaluate the "hair" on the CPU
    if (nBlocks > nBlocksAligned) {
        finishGravity(wp,tile->lstTile.nBlocks-nBlocksAligned,tile->lstTile.nInLast,tile->blk+nBlocksAligned);
        }
    return 1;
    }

extern "C"
int CUDA_queuePP(void *cudaCtx,workParticle *wp, ILPTILE tile, int bGravStep) {
    CUDACTX cuda = reinterpret_cast<CUDACTX>(cudaCtx);
    if(CUDA_queue< TB_THREADS, PP_WU, ILPTILE,ilpBlk<WIDTH> >(cuda,&cuda->nodePP,wp,tile,bGravStep,"PP")) {
        wp->dFlopSingleGPU += COST_FLOP_PP*wp->nP*(tile->lstTile.nBlocks*ILP_PART_PER_BLK  + tile->lstTile.nInLast);
        return 1;
        }
    return 0;
    }

extern "C"
int CUDA_queuePC(void *cudaCtx,workParticle *wp, ILCTILE tile, int bGravStep) {
    CUDACTX cuda = reinterpret_cast<CUDACTX>(cudaCtx);
    if (CUDA_queue< TB_THREADS, PC_WU, ILCTILE,ilcBlk<WIDTH> >(cuda,&cuda->nodePC,wp,tile,bGravStep,"PC")) {
        wp->dFlopSingleGPU += COST_FLOP_PC*wp->nP*(tile->lstTile.nBlocks*ILC_PART_PER_BLK  + tile->lstTile.nInLast);
        return 1;
        }
    return 0;
    }

