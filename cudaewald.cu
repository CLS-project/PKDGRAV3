#include "hip/hip_runtime.h"
/* -*- Mode: C++; tab-width: 8; indent-tabs-mode: nil; c-basic-offset: 4 -*- */
#ifdef HAVE_CONFIG_H
#include "config.h"
#endif
#include <time.h>
#ifdef HAVE_SYS_TIME_H
#include <sys/time.h>
#endif
#include <stdio.h>
#include "basetype.h"
#include "moments.h"
#include "cudautil.h"

#define ALIGN 64
#define MASK (ALIGN-1)

#define MAX_TOTAL_REPLICAS (7*7*7)

__constant__ struct EwaldVariables ew;
__constant__ float hx[MAX_TOTAL_REPLICAS];
__constant__ float hy[MAX_TOTAL_REPLICAS];
__constant__ float hz[MAX_TOTAL_REPLICAS];
__constant__ float hCfac[MAX_TOTAL_REPLICAS];
__constant__ float hSfac[MAX_TOTAL_REPLICAS];

__constant__ double Lx[MAX_TOTAL_REPLICAS];
__constant__ double Ly[MAX_TOTAL_REPLICAS];
__constant__ double Lz[MAX_TOTAL_REPLICAS];
__constant__ int bHole[MAX_TOTAL_REPLICAS];


/*
** nvcc -DHAVE_CONFIG_H --ptxas-options=-v -c  -I. -arch=sm_35 cudaewald.cu
** ptxas info    : 0 bytes gmem, 16948 bytes cmem[3]
** ptxas info    : Compiling entry function '_Z9cudaEwaldPdS_S_S_S_S_S_S_' for 'sm_35'
** ptxas info    : Function properties for _Z9cudaEwaldPdS_S_S_S_S_S_S_
**    32 bytes stack frame, 0 bytes spill stores, 0 bytes spill loads
** ptxas info    : Used 79 registers, 384 bytes cmem[0], 668 bytes cmem[2]
*/


/*
** threadIdx.x: all work on the same particle -- this is the warp size, i.e., 32
** blockIdx.x:  different particles. If y=z=1, then x can be anything, otherwise
**              the total number of particles is a block of x*y*z
*/

__global__ void cudaEwald(double *X,double *Y,double *Z,
    double *Xout, double *Yout, double *Zout, double *pPot,double *pdFlop) {
    int pidx = threadIdx.x + ALIGN*blockIdx.x;

    const double onethird = 1.0/3.0;
    double g0,g1,g2,g3,g4,g5,alphan;
    int i, bInHole;
    double tax, tay, taz, dPot, dFlop=0.0;
    const double rx = X[pidx] - ew.r[0];
    const double ry = Y[pidx] - ew.r[1];
    const double rz = Z[pidx] - ew.r[2];

    // the H-Loop
    float fx=rx, fy=ry, fz=rz;
    float fax=0, fay=0, faz=0, fPot=0;
    for( i=0; i<ew.nEwhLoop; ++i) {
	float hdotx,s,c,t;
	hdotx = hx[i]*fx + hy[i]*fy + hz[i]*fz;
	sincosf(hdotx,&s,&c);
	fPot += hCfac[i]*c + hSfac[i]*s;
	t = hCfac[i]*s - hSfac[i]*c;
	fax += hx[i]*t;
	fay += hy[i]*t;
	faz += hz[i]*t;
	}
    tax = fax;
    tay = fay;
    taz = faz;
    dPot = fPot;

    for(i=0; i<MAX_TOTAL_REPLICAS; ++i) {
        bInHole = bHole[i];
        const double x = rx + Lx[i];
        const double y = ry + Ly[i];
        const double z = rz + Lz[i];
        double r2 = x*x + y*y + z*z;
        if (r2 >= ew.fEwCut2 && !bInHole) continue;
        if (r2 < ew.fInner2) { /* Once, at most per particle */
            /*
             * For small r, series expand about
             * the origin to avoid errors caused
             * by cancellation of large terms.
             */
            alphan = ew.ka;
            r2 *= ew.alpha2;
            g0 = alphan*((1.0/3.0)*r2 - 1.0);
            alphan *= 2*ew.alpha2;
            g1 = alphan*((1.0/5.0)*r2 - (1.0/3.0));
            alphan *= 2*ew.alpha2;
            g2 = alphan*((1.0/7.0)*r2 - (1.0/5.0));
            alphan *= 2*ew.alpha2;
            g3 = alphan*((1.0/9.0)*r2 - (1.0/7.0));
            alphan *= 2*ew.alpha2;
            g4 = alphan*((1.0/11.0)*r2 - (1.0/9.0));
            alphan *= 2*ew.alpha2;
            g5 = alphan*((1.0/13.0)*r2 - (1.0/11.0));
            }
        else {
            const double dir = rsqrt(r2);
            const double dir2 = dir*dir;
            const double a = exp(-r2*ew.alpha2) * ew.ka*dir2;
            if (bInHole) g0 = -erf(ew.alpha*r2*dir);
            else         g0 = erfc(ew.alpha*r2*dir);
            g0 *= dir;
            g1 = g0*dir2 + a;
            alphan = 2*ew.alpha2;
            g2 = 3*g1*dir2 + alphan*a;
            alphan *= 2*ew.alpha2;
            g3 = 5*g2*dir2 + alphan*a;
            alphan *= 2*ew.alpha2;
            g4 = 7*g3*dir2 + alphan*a;
            alphan *= 2*ew.alpha2;
            g5 = 9*g4*dir2 + alphan*a;
            }

        dPot -= g0*ew.mom.m - g1*ew.Q2;

        double Q4mirx, Q4miry, Q4mirz;
        double Q3mirx, Q3miry, Q3mirz;

        const  double xx = 0.5*x*x;
        Q3mirx = ew.mom.xxx*xx;
        Q3miry = ew.mom.xxy*xx;
        Q3mirz = ew.mom.xxz*xx;
        const  double xxx = onethird*xx*x;
        Q4mirx = ew.mom.xxxx*xxx;
        Q4miry = ew.mom.xxxy*xxx;
        Q4mirz = ew.mom.xxxz*xxx;
        const  double xxy = xx*y;
        Q4mirx += ew.mom.xxxy*xxy;
        Q4miry += ew.mom.xxyy*xxy;
        Q4mirz += ew.mom.xxyz*xxy;
        const  double xxz = xx*z;
        Q4mirx += ew.mom.xxxz*xxz;
        Q4miry += ew.mom.xxyz*xxz;
        Q4mirz += ew.mom.xxzz*xxz;

        const  double yy = 0.5*y*y;
        Q3mirx += ew.mom.xyy*yy;
        Q3miry += ew.mom.yyy*yy;
        Q3mirz += ew.mom.yyz*yy;
        const  double xyy = yy*x;
        Q4mirx += ew.mom.xxyy*xyy;
        Q4miry += ew.mom.xyyy*xyy;
        Q4mirz += ew.mom.xyyz*xyy;
        const  double yyy = onethird*yy*y;
        Q4mirx += ew.mom.xyyy*yyy;
        Q4miry += ew.mom.yyyy*yyy;
        Q4mirz += ew.mom.yyyz*yyy;
        const  double yyz = yy*z;
        Q4mirx += ew.mom.xyyz*yyz;
        Q4miry += ew.mom.yyyz*yyz;
        Q4mirz += ew.mom.yyzz*yyz;

        const  double xy = x*y;
        Q3mirx += ew.mom.xxy*xy;
        Q3miry += ew.mom.xyy*xy;
        Q3mirz += ew.mom.xyz*xy;
        const  double xyz = xy*z;
        Q4mirx += ew.mom.xxyz*xyz;
        Q4miry += ew.mom.xyyz*xyz;
        Q4mirz += ew.mom.xyzz*xyz;

        const  double zz = 0.5*z*z;
        Q3mirx += ew.mom.xzz*zz;
        Q3miry += ew.mom.yzz*zz;
        Q3mirz += ew.mom.zzz*zz;
        const  double xzz = zz*x;
        Q4mirx += ew.mom.xxzz*xzz;
        Q4miry += ew.mom.xyzz*xzz;
        Q4mirz += ew.mom.xzzz*xzz;
        const  double yzz = zz*y;
        Q4mirx += ew.mom.xyzz*yzz;
        Q4miry += ew.mom.yyzz*yzz;
        Q4mirz += ew.mom.yzzz*yzz;
        const  double zzz = onethird*zz*z;
        Q4mirx += ew.mom.xzzz*zzz;
        Q4miry += ew.mom.yzzz*zzz;
        Q4mirz += ew.mom.zzzz*zzz;

        tax += g4*Q4mirx;
        tay += g4*Q4miry;
        taz += g4*Q4mirz;
        const double Q4mir = 0.25*(Q4mirx*x + Q4miry*y + Q4mirz*z);
        dPot -= g4*Q4mir;

        const  double xz = x*z;
        Q3mirx += ew.mom.xxz*xz;
        Q3miry += ew.mom.xyz*xz;
        Q3mirz += ew.mom.xzz*xz;

        const  double yz = y*z;
        Q3mirx += ew.mom.xyz*yz;
        Q3miry += ew.mom.yyz*yz;
        Q3mirz += ew.mom.yzz*yz;

        const double Q4x = ew.Q4xx*x + ew.Q4xy*y + ew.Q4xz*z;
        const double Q4y = ew.Q4xy*x + ew.Q4yy*y + ew.Q4yz*z;
        const double Q4z = ew.Q4xz*x + ew.Q4yz*y + ew.Q4zz*z;
        const double Q3mir = onethird*(Q3mirx*x + Q3miry*y + Q3mirz*z) - 0.5*(Q4x*x + Q4y*y + Q4z*z);
        dPot -= g3*Q3mir;
        tax += g3*(Q3mirx - Q4x);
        tay += g3*(Q3miry - Q4y);
        taz += g3*(Q3mirz - Q4z);

        const double Q2mirx = ew.mom.xx*x + ew.mom.xy*y + ew.mom.xz*z;
        const double Q2miry = ew.mom.xy*x + ew.mom.yy*y + ew.mom.yz*z;
        const double Q2mirz = ew.mom.xz*x + ew.mom.yz*y + ew.mom.zz*z;
        const double Q2mir = 0.5*(Q2mirx*x + Q2miry*y + Q2mirz*z) - (ew.Q3x*x + ew.Q3y*y + ew.Q3z*z) + ew.Q4;
        dPot -= g2*Q2mir;
        tax += g2*(Q2mirx - ew.Q3x);
        tay += g2*(Q2miry - ew.Q3y);
        taz += g2*(Q2mirz - ew.Q3z);

        const double Qta = g1*ew.mom.m - g2*ew.Q2 + g3*Q2mir + g4*Q3mir + g5*Q4mir;
        tax -= x*Qta;
        tay -= y*Qta;
        taz -= z*Qta;
        dFlop += COST_FLOP_EWALD;
	}

/*    dFlop += COST_FLOP_HLOOP * ew.nEwhLoop;*/ /* Accounted for outside */
    Xout[pidx] = tax;
    Yout[pidx] = tay;
    Zout[pidx] = taz;
    pPot[pidx] = dPot;
    pdFlop[pidx] = dFlop;
    }

/* If this returns an error, then the caller must attempt recovery or abort */
hipError_t cuda_setup_ewald(CUDACTX cuda) {
    if (cuda->ewIn && cuda->ewt) {
        double start = CUDA_getTime();
        CUDA_RETURN(hipMemcpyToSymbolAsync,(ew,cuda->ewIn,sizeof(ew),0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(hx,cuda->ewt->hx.f,sizeof(float)*cuda->ewIn->nEwhLoop,0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(hy,cuda->ewt->hy.f,sizeof(float)*cuda->ewIn->nEwhLoop,0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(hz,cuda->ewt->hz.f,sizeof(float)*cuda->ewIn->nEwhLoop,0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(hCfac,cuda->ewt->hCfac.f,sizeof(float)*cuda->ewIn->nEwhLoop,0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(hSfac,cuda->ewt->hSfac.f,sizeof(float)*cuda->ewIn->nEwhLoop,0,hipMemcpyHostToDevice,cuda->streamEwald));
// Time(%)      Time     Calls       Avg       Min       Max  Name
// 14.93%  1.47255s       413  3.5655ms  2.6458ms  3.9733ms  cudaEwald(double*, double*, double*, double*, double*, double*, double*, d
        double dLx[MAX_TOTAL_REPLICAS];
        double dLy[MAX_TOTAL_REPLICAS];
        double dLz[MAX_TOTAL_REPLICAS];
        int ibHole[MAX_TOTAL_REPLICAS];
        int i=0, ix, iy, iz;
        for(ix=-3; ix<=3; ++ix) {
            for(iy=-3; iy<=3; ++iy) {
                for(iz=-3; iz<=3; ++iz) {
                    ibHole[i] = (abs(ix) <= cuda->ewIn->nReps && abs(iy) <= cuda->ewIn->nReps && abs(iz) <= cuda->ewIn->nReps);
                    dLx[i] = cuda->ewIn->Lbox * ix;
                    dLy[i] = cuda->ewIn->Lbox * iy;
                    dLz[i] = cuda->ewIn->Lbox * iz;
                    ++i;
                    }
                }
            }
        CUDA_RETURN(hipMemcpyToSymbolAsync,(Lx,dLx,sizeof(Lx),0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(Ly,dLy,sizeof(Ly),0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(Lz,dLz,sizeof(Lz),0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(bHole,ibHole,sizeof(bHole),0,hipMemcpyHostToDevice,cuda->streamEwald));


#ifdef USE_CUDA_EVENTS
        CUDA_RETURN(hipEventRecord,(cuda->eventEwald,cuda->streamEwald));
#endif
        hipError_t rc;
        do {
#ifdef USE_CUDA_EVENTS
            rc = hipEventQuery(cuda->eventEwald);
#else
            rc = hipStreamQuery(cuda->streamEwald);
#endif
            switch(rc) {
            case hipSuccess:
            case hipErrorNotReady:
                break;
            default:
                return rc;
                }
            if (CUDA_getTime() - start > 1.0) {
                return hipErrorLaunchTimeOut;
                }
            } while (rc!=hipSuccess);
        }
    return hipSuccess;
    }

extern "C"
void cudaEwaldInit(void *cudaCtx, struct EwaldVariables *ewIn, EwaldTable *ewt ) {
    CUDACTX cuda = reinterpret_cast<CUDACTX>(cudaCtx);
    cuda->ewIn = ewIn;
    cuda->ewt = ewt;
    if (cuda->iCore==0) {
        hipError_t ec = cuda_setup_ewald(cuda);
        if (ec != hipSuccess) CUDA_attempt_recovery(cuda,ec);
        }
    }

/* If this returns an error, then the caller must attempt recovery or abort */
extern "C"
int CUDAinitWorkEwald( void *ve, void *vwork ) {
    workEwald *e = reinterpret_cast<workEwald *>(ve);
    CUDAwqNode *work = reinterpret_cast<CUDAwqNode *>(vwork);
    double *pHostBufFromGPU  = reinterpret_cast<double *>(work->pHostBufFromGPU);
    double *pHostBufToGPU    = reinterpret_cast<double *>(work->pHostBufToGPU);
    double *pCudaBufIn = reinterpret_cast<double *>(work->pCudaBufIn);
    double *pCudaBufOut = reinterpret_cast<double *>(work->pCudaBufOut);
    double *X, *Y, *Z;
    double *cudaX, *cudaY, *cudaZ, *cudaXout, *cudaYout, *cudaZout, *cudaPot, *cudaFlop;
    int align, i;

    align = (e->nP+MASK)&~MASK; /* Warp align the memory buffers */
    X       = pHostBufToGPU + 0*align;
    Y       = pHostBufToGPU + 1*align;
    Z       = pHostBufToGPU + 2*align;
    cudaX   = pCudaBufIn + 0*align;
    cudaY   = pCudaBufIn + 1*align;
    cudaZ   = pCudaBufIn + 2*align;
    cudaXout= pCudaBufOut + 0*align;
    cudaYout= pCudaBufOut + 1*align;
    cudaZout= pCudaBufOut + 2*align;
    cudaPot = pCudaBufOut + 3*align;
    cudaFlop= pCudaBufOut + 4*align;

    dim3 dimBlock( ALIGN, 1 );
    dim3 dimGrid( align/ALIGN, 1,1 );
    for(i=0; i<e->nP; ++i) {
        const workParticle *wp = e->ppWorkPart[i];
	const int wi = e->piWorkPart[i];
	const PINFOIN *in = &wp->pInfoIn[wi];
	X[i] = wp->c[0] + in->r[0];
	Y[i] = wp->c[1] + in->r[1];
	Z[i] = wp->c[2] + in->r[2];
	}
    for(;i<align;++i) X[i]=Y[i]=Z[i] = 100;

    // copy data directly to device memory
    CUDA_RETURN(hipMemcpyAsync,(pCudaBufIn, pHostBufToGPU, align*3*sizeof(double),
	    hipMemcpyHostToDevice, work->stream));
    cudaEwald<<<dimGrid, dimBlock, 0, work->stream>>>(cudaX,cudaY,cudaZ,cudaXout,cudaYout,cudaZout,cudaPot,cudaFlop);
    CUDA_RETURN(hipMemcpyAsync,(pHostBufFromGPU, pCudaBufOut, align*5*sizeof(double),
            hipMemcpyDeviceToHost, work->stream));
#ifdef USE_CUDA_EVENTS
    CUDA_RETURN(hipEventRecord,(work->event,work->stream));
#endif

    return hipSuccess;
    }

extern "C"
void pkdAccumulateCUDA(void * pkd,workEwald *we,double *pax,double *pay,double *paz,double *pot,double *pdFlop);


extern "C"
int CUDAcheckWorkEwald( void *ve, void *vwork ) {
    workEwald *e = reinterpret_cast<workEwald *>(ve);
    CUDAwqNode *work = reinterpret_cast<CUDAwqNode *>(vwork);
    double *pHostBuf = reinterpret_cast<double *>(work->pHostBufFromGPU);
    double *X, *Y, *Z, *pPot, *pdFlop;
    int align;

    align = (e->nP+MASK)&~MASK; /* As above! Warp align the memory buffers */
    X       = pHostBuf + 0*align;
    Y       = pHostBuf + 1*align;
    Z       = pHostBuf + 2*align;
    pPot    = pHostBuf + 3*align;
    pdFlop  = pHostBuf + 4*align;
    pkdAccumulateCUDA(e->pkd,e,X,Y,Z,pPot,pdFlop);
    free(e->ppWorkPart);
    free(e->piWorkPart);
    free(e);
    return 0;

    }
