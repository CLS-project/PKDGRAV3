#include "hip/hip_runtime.h"
/* -*- Mode: C++; tab-width: 8; indent-tabs-mode: nil; c-basic-offset: 4 -*- */
#ifdef HAVE_CONFIG_H
#include "config.h"
#endif
#include <time.h>
#ifdef HAVE_SYS_TIME_H
#include <sys/time.h>
#endif
#include <stdio.h>
#include "basetype.h"
#include "moments.h"
#include "cudautil.h"

#define ALIGN 64
#define MASK (ALIGN-1)

#define MAX_TOTAL_REPLICAS (7*7*7)

__constant__ struct EwaldVariables ew;
__constant__ float hx[MAX_TOTAL_REPLICAS];
__constant__ float hy[MAX_TOTAL_REPLICAS];
__constant__ float hz[MAX_TOTAL_REPLICAS];
__constant__ float hCfac[MAX_TOTAL_REPLICAS];
__constant__ float hSfac[MAX_TOTAL_REPLICAS];

__constant__ double Lx[MAX_TOTAL_REPLICAS];
__constant__ double Ly[MAX_TOTAL_REPLICAS];
__constant__ double Lz[MAX_TOTAL_REPLICAS];
__constant__ int bHole[MAX_TOTAL_REPLICAS];


/*
** nvcc -DHAVE_CONFIG_H --ptxas-options=-v -c  -I. -arch=sm_35 cudaewald.cu
** ptxas info    : 0 bytes gmem, 16948 bytes cmem[3]
** ptxas info    : Compiling entry function '_Z9cudaEwaldPdS_S_S_S_S_S_S_' for 'sm_35'
** ptxas info    : Function properties for _Z9cudaEwaldPdS_S_S_S_S_S_S_
**    32 bytes stack frame, 0 bytes spill stores, 0 bytes spill loads
** ptxas info    : Used 79 registers, 384 bytes cmem[0], 668 bytes cmem[2]
*/


/*
** threadIdx.x: all work on the same particle -- this is the warp size, i.e., 32
** blockIdx.x:  different particles. If y=z=1, then x can be anything, otherwise
**              the total number of particles is a block of x*y*z
**
** We are allowed 16 resident blocks so this corresponds to 512 threads per SM.
** This is fine because we are actually close to shared memory limited:
**   bValid[512] = 2052 bytes
**   We have 48k per SM so around 23 active thread blocks (but we use only 16)
**   compute 5.0: 32 thread blocks, but 64K shared gives us ~ 31 resident.
*/

__global__ void cudaEwald(double *X,double *Y,double *Z,
    double *Xout, double *Yout, double *Zout, double *pPot,double *pdFlop) {
    const double onethird = 1.0/3.0;
    double g0,g1,g2,g3,g4,g5,alphan;
    int i, bInHole;
    int pidx = threadIdx.x + ALIGN*blockIdx.x;
    double tax = 0.0, tay = 0.0, taz = 0.0, dPot=0.0, dFlop=0.0;

    const double rx = X[pidx] - ew.r[0];
    const double ry = Y[pidx] - ew.r[1];
    const double rz = Z[pidx] - ew.r[2];
    for(i=0; i<MAX_TOTAL_REPLICAS; ++i) {
        bInHole = bHole[i];
        const double x = rx + Lx[i];
        const double y = ry + Ly[i];
        const double z = rz + Lz[i];
        double r2 = x*x + y*y + z*z;
        if (r2 >= ew.fEwCut2 && !bInHole) continue;
        if (r2 < ew.fInner2) { /* Once, at most per particle */
            /*
             * For small r, series expand about
             * the origin to avoid errors caused
             * by cancellation of large terms.
             */
            alphan = ew.ka;
            r2 *= ew.alpha2;
            g0 = alphan*((1.0/3.0)*r2 - 1.0);
            alphan *= 2*ew.alpha2;
            g1 = alphan*((1.0/5.0)*r2 - (1.0/3.0));
            alphan *= 2*ew.alpha2;
            g2 = alphan*((1.0/7.0)*r2 - (1.0/5.0));
            alphan *= 2*ew.alpha2;
            g3 = alphan*((1.0/9.0)*r2 - (1.0/7.0));
            alphan *= 2*ew.alpha2;
            g4 = alphan*((1.0/11.0)*r2 - (1.0/9.0));
            alphan *= 2*ew.alpha2;
            g5 = alphan*((1.0/13.0)*r2 - (1.0/11.0));
            }
        else {
            const double dir = rsqrt(r2);
            const double dir2 = dir*dir;
            const double a = exp(-r2*ew.alpha2) * ew.ka*dir2;
            if (bInHole) g0 = -erf(ew.alpha*r2*dir);
            else         g0 = erfc(ew.alpha*r2*dir);
            g0 *= dir;
            g1 = g0*dir2 + a;
            alphan = 2*ew.alpha2;
            g2 = 3*g1*dir2 + alphan*a;
            alphan *= 2*ew.alpha2;
            g3 = 5*g2*dir2 + alphan*a;
            alphan *= 2*ew.alpha2;
            g4 = 7*g3*dir2 + alphan*a;
            alphan *= 2*ew.alpha2;
            g5 = 9*g4*dir2 + alphan*a;
            }

        dPot -= g0*ew.mom.m - g1*ew.Q2;

        const  double xx = 0.5*x*x;
        const  double xxx = onethird*xx*x;
        const  double xxy = xx*y;
        const  double xxz = xx*z;
        const  double yy = 0.5*y*y;
        const  double yyy = onethird*yy*y;
        const  double xyy = yy*x;
        const  double yyz = yy*z;
        const  double zz = 0.5*z*z;
        const  double zzz = onethird*zz*z;
        const  double xzz = zz*x;
        const  double yzz = zz*y;
        const  double xy = x*y;
        const  double xyz = xy*z;
        const  double xz = x*z;
        const  double yz = y*z;

        const double Q4mirx = ew.mom.xxxx*xxx + ew.mom.xxxy*xxy + ew.mom.xxxz*xxz + ew.mom.xxyy*xyy + ew.mom.xxyz*xyz +
            ew.mom.xxzz*xzz + ew.mom.xyyy*yyy + ew.mom.xyyz*yyz + ew.mom.xyzz*yzz + ew.mom.xzzz*zzz;
        const double Q4miry = ew.mom.xxxy*xxx + ew.mom.xxyy*xxy + ew.mom.xxyz*xxz + ew.mom.xyyy*xyy + ew.mom.xyyz*xyz +
            ew.mom.xyzz*xzz + ew.mom.yyyy*yyy + ew.mom.yyyz*yyz + ew.mom.yyzz*yzz + ew.mom.yzzz*zzz;
        const double Q4mirz = ew.mom.xxxz*xxx + ew.mom.xxyz*xxy + ew.mom.xxzz*xxz + ew.mom.xyyz*xyy + ew.mom.xyzz*xyz +
            ew.mom.xzzz*xzz + ew.mom.yyyz*yyy + ew.mom.yyzz*yyz + ew.mom.yzzz*yzz + ew.mom.zzzz*zzz;
        const double Q4mir = 0.25*(Q4mirx*x + Q4miry*y + Q4mirz*z);
        dPot -= g4*Q4mir;
        tax += g4*Q4mirx;
        tay += g4*Q4miry;
        taz += g4*Q4mirz;

        const double Q3mirx = ew.mom.xxx*xx + ew.mom.xxy*xy + ew.mom.xxz*xz + ew.mom.xyy*yy + ew.mom.xyz*yz + ew.mom.xzz*zz;
        const double Q3miry = ew.mom.xxy*xx + ew.mom.xyy*xy + ew.mom.xyz*xz + ew.mom.yyy*yy + ew.mom.yyz*yz + ew.mom.yzz*zz;
        const double Q3mirz = ew.mom.xxz*xx + ew.mom.xyz*xy + ew.mom.xzz*xz + ew.mom.yyz*yy + ew.mom.yzz*yz + ew.mom.zzz*zz;
        const double Q4x = ew.Q4xx*x + ew.Q4xy*y + ew.Q4xz*z;
        const double Q4y = ew.Q4xy*x + ew.Q4yy*y + ew.Q4yz*z;
        const double Q4z = ew.Q4xz*x + ew.Q4yz*y + ew.Q4zz*z;
        const double Q3mir = onethird*(Q3mirx*x + Q3miry*y + Q3mirz*z) - 0.5*(Q4x*x + Q4y*y + Q4z*z);
        dPot -= g3*Q3mir;
        tax += g3*(Q3mirx - Q4x);
        tay += g3*(Q3miry - Q4y);
        taz += g3*(Q3mirz - Q4z);

        const double Q2mirx = ew.mom.xx*x + ew.mom.xy*y + ew.mom.xz*z;
        const double Q2miry = ew.mom.xy*x + ew.mom.yy*y + ew.mom.yz*z;
        const double Q2mirz = ew.mom.xz*x + ew.mom.yz*y + ew.mom.zz*z;
        const double Q2mir = 0.5*(Q2mirx*x + Q2miry*y + Q2mirz*z) - (ew.Q3x*x + ew.Q3y*y + ew.Q3z*z) + ew.Q4;
        dPot -= g2*Q2mir;
        tax += g2*(Q2mirx - ew.Q3x);
        tay += g2*(Q2miry - ew.Q3y);
        taz += g2*(Q2mirz - ew.Q3z);

//                dPot -= g0*ew.mom.m - g1*ew.Q2 + g2*Q2mir + g3*Q3mir + g4*Q4mir;

        const double Qta = g1*ew.mom.m - g2*ew.Q2 + g3*Q2mir + g4*Q3mir + g5*Q4mir;
        tax -= x*Qta;
        tay -= y*Qta;
        taz -= z*Qta;
        dFlop += COST_FLOP_EWALD;
	}

    // the H-Loop
    float fx=rx, fy=ry, fz=rz;
    float fax=0, fay=0, faz=0, fPot=0;
    for( i=0; i<ew.nEwhLoop; ++i) {
	float hdotx,s,c,t;
	hdotx = hx[i]*fx + hy[i]*fy + hz[i]*fz;
	sincosf(hdotx,&s,&c);
	fPot += hCfac[i]*c + hSfac[i]*s;
	t = hCfac[i]*s - hSfac[i]*c;
	fax += hx[i]*t;
	fay += hy[i]*t;
	faz += hz[i]*t;
	}
/*    dFlop += COST_FLOP_HLOOP * ew.nEwhLoop;*/ /* Accounted for outside */
    Xout[pidx] = tax + fax;
    Yout[pidx] = tay + fay;
    Zout[pidx] = taz + faz;
    pPot[pidx] = dPot + fPot;
    pdFlop[pidx] = dFlop;
    }

/* If this returns an error, then the caller must attempt recovery or abort */
hipError_t cuda_setup_ewald(CUDACTX cuda) {
    if (cuda->ewIn && cuda->ewt) {
        double start = CUDA_getTime();
        CUDA_RETURN(hipMemcpyToSymbolAsync,(ew,cuda->ewIn,sizeof(ew),0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(hx,cuda->ewt->hx.f,sizeof(float)*cuda->ewIn->nEwhLoop,0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(hy,cuda->ewt->hy.f,sizeof(float)*cuda->ewIn->nEwhLoop,0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(hz,cuda->ewt->hz.f,sizeof(float)*cuda->ewIn->nEwhLoop,0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(hCfac,cuda->ewt->hCfac.f,sizeof(float)*cuda->ewIn->nEwhLoop,0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(hSfac,cuda->ewt->hSfac.f,sizeof(float)*cuda->ewIn->nEwhLoop,0,hipMemcpyHostToDevice,cuda->streamEwald));
// Time(%)      Time     Calls       Avg       Min       Max  Name
// 14.93%  1.47255s       413  3.5655ms  2.6458ms  3.9733ms  cudaEwald(double*, double*, double*, double*, double*, double*, double*, d
        double dLx[MAX_TOTAL_REPLICAS];
        double dLy[MAX_TOTAL_REPLICAS];
        double dLz[MAX_TOTAL_REPLICAS];
        int ibHole[MAX_TOTAL_REPLICAS];
        int i=0, ix, iy, iz;
        for(ix=-3; ix<=3; ++ix) {
            for(iy=-3; iy<=3; ++iy) {
                for(iz=-3; iz<=3; ++iz) {
                    ibHole[i] = (abs(ix) <= cuda->ewIn->nReps && abs(iy) <= cuda->ewIn->nReps && abs(iz) <= cuda->ewIn->nReps);
                    dLx[i] = cuda->ewIn->Lbox * ix;
                    dLy[i] = cuda->ewIn->Lbox * iy;
                    dLz[i] = cuda->ewIn->Lbox * iz;
                    ++i;
                    }
                }
            }
        CUDA_RETURN(hipMemcpyToSymbolAsync,(Lx,dLx,sizeof(Lx),0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(Ly,dLy,sizeof(Ly),0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(Lz,dLz,sizeof(Lz),0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(bHole,ibHole,sizeof(bHole),0,hipMemcpyHostToDevice,cuda->streamEwald));


#ifdef USE_CUDA_EVENTS
        CUDA_RETURN(hipEventRecord,(cuda->eventEwald,cuda->streamEwald));
#endif
        hipError_t rc;
        do {
#ifdef USE_CUDA_EVENTS
            rc = hipEventQuery(cuda->eventEwald);
#else
            rc = hipStreamQuery(cuda->streamEwald);
#endif
            switch(rc) {
            case hipSuccess:
            case hipErrorNotReady:
                break;
            default:
                return rc;
                }
            if (CUDA_getTime() - start > 1.0) {
                return hipErrorLaunchTimeOut;
                }
            } while (rc!=hipSuccess);
        }
    return hipSuccess;
    }

extern "C"
void cudaEwaldInit(void *cudaCtx, struct EwaldVariables *ewIn, EwaldTable *ewt ) {
    CUDACTX cuda = reinterpret_cast<CUDACTX>(cudaCtx);
    cuda->ewIn = ewIn;
    cuda->ewt = ewt;
    if (cuda->iCore==0) {
        hipError_t ec = cuda_setup_ewald(cuda);
        if (ec != hipSuccess) CUDA_attempt_recovery(cuda,ec);
        }
    }

/* If this returns an error, then the caller must attempt recovery or abort */
extern "C"
int CUDAinitWorkEwald( void *ve, void *vwork ) {
    workEwald *e = reinterpret_cast<workEwald *>(ve);
    CUDAwqNode *work = reinterpret_cast<CUDAwqNode *>(vwork);
    double *pHostBufFromGPU  = reinterpret_cast<double *>(work->pHostBufFromGPU);
    double *pHostBufToGPU    = reinterpret_cast<double *>(work->pHostBufToGPU);
    double *pCudaBufIn = reinterpret_cast<double *>(work->pCudaBufIn);
    double *pCudaBufOut = reinterpret_cast<double *>(work->pCudaBufOut);
    double *X, *Y, *Z;
    double *cudaX, *cudaY, *cudaZ, *cudaXout, *cudaYout, *cudaZout, *cudaPot, *cudaFlop;
    int align, i;

    align = (e->nP+MASK)&~MASK; /* Warp align the memory buffers */
    X       = pHostBufToGPU + 0*align;
    Y       = pHostBufToGPU + 1*align;
    Z       = pHostBufToGPU + 2*align;
    cudaX   = pCudaBufIn + 0*align;
    cudaY   = pCudaBufIn + 1*align;
    cudaZ   = pCudaBufIn + 2*align;
    cudaXout= pCudaBufOut + 0*align;
    cudaYout= pCudaBufOut + 1*align;
    cudaZout= pCudaBufOut + 2*align;
    cudaPot = pCudaBufOut + 3*align;
    cudaFlop= pCudaBufOut + 4*align;

    dim3 dimBlock( ALIGN, 1 );
    dim3 dimGrid( align/ALIGN, 1,1 );
    for(i=0; i<e->nP; ++i) {
        const workParticle *wp = e->ppWorkPart[i];
	const int wi = e->piWorkPart[i];
	const PINFOIN *in = &wp->pInfoIn[wi];
	X[i] = wp->c[0] + in->r[0];
	Y[i] = wp->c[1] + in->r[1];
	Z[i] = wp->c[2] + in->r[2];
	}
    for(;i<align;++i) X[i]=Y[i]=Z[i] = 100;

    // copy data directly to device memory
    CUDA_RETURN(hipMemcpyAsync,(pCudaBufIn, pHostBufToGPU, align*3*sizeof(double),
	    hipMemcpyHostToDevice, work->stream));
    cudaEwald<<<dimGrid, dimBlock, 0, work->stream>>>(cudaX,cudaY,cudaZ,cudaXout,cudaYout,cudaZout,cudaPot,cudaFlop);
    CUDA_RETURN(hipMemcpyAsync,(pHostBufFromGPU, pCudaBufOut, align*5*sizeof(double),
            hipMemcpyDeviceToHost, work->stream));
#ifdef USE_CUDA_EVENTS
    CUDA_RETURN(hipEventRecord,(work->event,work->stream));
#endif

    return hipSuccess;
    }

extern "C"
void pkdAccumulateCUDA(void * pkd,workEwald *we,double *pax,double *pay,double *paz,double *pot,double *pdFlop);


extern "C"
int CUDAcheckWorkEwald( void *ve, void *vwork ) {
    workEwald *e = reinterpret_cast<workEwald *>(ve);
    CUDAwqNode *work = reinterpret_cast<CUDAwqNode *>(vwork);
    double *pHostBuf = reinterpret_cast<double *>(work->pHostBufFromGPU);
    double *X, *Y, *Z, *pPot, *pdFlop;
    int align;

    align = (e->nP+MASK)&~MASK; /* As above! Warp align the memory buffers */
    X       = pHostBuf + 0*align;
    Y       = pHostBuf + 1*align;
    Z       = pHostBuf + 2*align;
    pPot    = pHostBuf + 3*align;
    pdFlop  = pHostBuf + 4*align;
    pkdAccumulateCUDA(e->pkd,e,X,Y,Z,pPot,pdFlop);
    free(e->ppWorkPart);
    free(e->piWorkPart);
    free(e);
    return 0;

    }
