#include "hip/hip_runtime.h"
/* -*- Mode: C++; tab-width: 8; indent-tabs-mode: nil; c-basic-offset: 4 -*- */
#ifdef HAVE_CONFIG_H
#include "config.h"
#endif
#include <time.h>
#ifdef HAVE_SYS_TIME_H
#include <sys/time.h>
#endif
#include <stdio.h>
#include "basetype.h"
#include "moments.h"
#include "cudautil.h"

#define ALIGN 64
#define MASK (ALIGN-1)

#define MAX_TOTAL_REPLICAS (7*7*7)

__constant__ struct EwaldVariables ew;
__constant__ float hx[MAX_TOTAL_REPLICAS];
__constant__ float hy[MAX_TOTAL_REPLICAS];
__constant__ float hz[MAX_TOTAL_REPLICAS];
__constant__ float hCfac[MAX_TOTAL_REPLICAS];
__constant__ float hSfac[MAX_TOTAL_REPLICAS];

__constant__ momFloat Lx[MAX_TOTAL_REPLICAS];
__constant__ momFloat Ly[MAX_TOTAL_REPLICAS];
__constant__ momFloat Lz[MAX_TOTAL_REPLICAS];
__constant__ int bHole[MAX_TOTAL_REPLICAS];


/*
** nvcc -DHAVE_CONFIG_H --ptxas-options=-v -c  -I. -arch=sm_35 cudaewald.cu
** ptxas info    : 0 bytes gmem, 16948 bytes cmem[3]
** ptxas info    : Compiling entry function '_Z9cudaEwaldPdS_S_S_S_S_S_S_' for 'sm_35'
** ptxas info    : Function properties for _Z9cudaEwaldPdS_S_S_S_S_S_S_
**    32 bytes stack frame, 0 bytes spill stores, 0 bytes spill loads
** ptxas info    : Used 79 registers, 384 bytes cmem[0], 668 bytes cmem[2]
*/


/*
** threadIdx.x: all work on the same particle -- this is the warp size, i.e., 32
** blockIdx.x:  different particles. If y=z=1, then x can be anything, otherwise
**              the total number of particles is a block of x*y*z
*/

__global__ void cudaEwald(momFloat *X,momFloat *Y,momFloat *Z,
    momFloat *Xout, momFloat *Yout, momFloat *Zout, momFloat *pPot,momFloat *pdFlop) {
    int pidx = threadIdx.x + ALIGN*blockIdx.x;
    const momFloat f_1_2 = 1.0 / 2.0;
    const momFloat f_1_3 = 1.0 / 3.0;
    const momFloat f_1_4 = 1.0 / 4.0;
    const momFloat f_1_5 = 1.0 / 5.0;
    const momFloat f_1_7 = 1.0 / 7.0;
    const momFloat f_1_9 = 1.0 / 9.0;
    const momFloat f_1_11 = 1.0 / 11.0;
    const momFloat f_1_13 = 1.0 / 13.0;
    momFloat g0, g1, g2, g3, g4, g5, alphan;
    int i, bInHole;
    momFloat tax, tay, taz, dPot, dFlop=0.0;
    const momFloat rx = X[pidx] - ew.r[0];
    const momFloat ry = Y[pidx] - ew.r[1];
    const momFloat rz = Z[pidx] - ew.r[2];

    // the H-Loop
    float fx=rx, fy=ry, fz=rz;
    float fax=0, fay=0, faz=0, fPot=0;
    for( i=0; i<ew.nEwhLoop; ++i) {
	float hdotx,s,c,t;
	hdotx = hx[i]*fx + hy[i]*fy + hz[i]*fz;
	sincosf(hdotx,&s,&c);
	fPot += hCfac[i]*c + hSfac[i]*s;
	t = hCfac[i]*s - hSfac[i]*c;
	fax += hx[i]*t;
	fay += hy[i]*t;
	faz += hz[i]*t;
	}
    tax = fax;
    tay = fay;
    taz = faz;
    dPot = fPot;

    for(i=0; i<MAX_TOTAL_REPLICAS; ++i) {
        bInHole = bHole[i];
        const momFloat x = rx + Lx[i];
        const momFloat y = ry + Ly[i];
        const momFloat z = rz + Lz[i];
        momFloat r2 = x*x + y*y + z*z;
        if (r2 >= ew.fEwCut2 && !bInHole) continue;
        if (r2 < ew.fInner2) { /* Once, at most per particle */
            /*
             * For small r, series expand about
             * the origin to avoid errors caused
             * by cancellation of large terms.
             */
            alphan = ew.ka;
            r2 *= ew.alpha2;
            g0 = alphan*(f_1_3*r2 - 1);
            alphan *= 2*ew.alpha2;
	    g1 = alphan*(f_1_5*r2 - f_1_3);
            alphan *= 2*ew.alpha2;
	    g2 = alphan*(f_1_7*r2 - f_1_5);
            alphan *= 2*ew.alpha2;
	    g3 = alphan*(f_1_9*r2 - f_1_7);
            alphan *= 2*ew.alpha2;
	    g4 = alphan*(f_1_11*r2 - f_1_9);
            alphan *= 2*ew.alpha2;
	    g5 = alphan*(f_1_13*r2 - f_1_11);
            }
        else {
	    const momFloat dir = rsqrt(r2);
	    const momFloat dir2 = dir*dir;
	    const momFloat a = exp(-r2*ew.alpha2) * ew.ka*dir2;
	    if (bInHole) g0 = -erf(ew.alpha*r2*dir);
	    else         g0 = erfc(ew.alpha*r2*dir);
	    g0 *= dir;
            g1 = g0*dir2 + a;
            alphan = 2*ew.alpha2;
            g2 = 3*g1*dir2 + alphan*a;
            alphan *= 2*ew.alpha2;
            g3 = 5*g2*dir2 + alphan*a;
            alphan *= 2*ew.alpha2;
            g4 = 7*g3*dir2 + alphan*a;
            alphan *= 2*ew.alpha2;
            g5 = 9*g4*dir2 + alphan*a;
            }

        dPot -= g0*ew.mom.m - g1*ew.Q2;

        momFloat Q4mirx, Q4miry, Q4mirz;
        momFloat Q3mirx, Q3miry, Q3mirz;

	const  momFloat xx = f_1_2*x*x;
        Q3mirx = ew.mom.xxx*xx;
        Q3miry = ew.mom.xxy*xx;
        Q3mirz = ew.mom.xxz*xx;
	const  momFloat xxx = f_1_3*xx*x;
        Q4mirx = ew.mom.xxxx*xxx;
        Q4miry = ew.mom.xxxy*xxx;
        Q4mirz = ew.mom.xxxz*xxx;
        const  momFloat xxy = xx*y;
        Q4mirx += ew.mom.xxxy*xxy;
        Q4miry += ew.mom.xxyy*xxy;
        Q4mirz += ew.mom.xxyz*xxy;
        const  momFloat xxz = xx*z;
        Q4mirx += ew.mom.xxxz*xxz;
        Q4miry += ew.mom.xxyz*xxz;
        Q4mirz += ew.mom.xxzz*xxz;

	const  momFloat yy = f_1_2*y*y;
        Q3mirx += ew.mom.xyy*yy;
        Q3miry += ew.mom.yyy*yy;
        Q3mirz += ew.mom.yyz*yy;
        const  momFloat xyy = yy*x;
        Q4mirx += ew.mom.xxyy*xyy;
        Q4miry += ew.mom.xyyy*xyy;
        Q4mirz += ew.mom.xyyz*xyy;
	const  momFloat yyy = f_1_3*yy*y;
        Q4mirx += ew.mom.xyyy*yyy;
        Q4miry += ew.mom.yyyy*yyy;
        Q4mirz += ew.mom.yyyz*yyy;
        const  momFloat yyz = yy*z;
        Q4mirx += ew.mom.xyyz*yyz;
        Q4miry += ew.mom.yyyz*yyz;
        Q4mirz += ew.mom.yyzz*yyz;

        const  momFloat xy = x*y;
        Q3mirx += ew.mom.xxy*xy;
        Q3miry += ew.mom.xyy*xy;
        Q3mirz += ew.mom.xyz*xy;
        const  momFloat xyz = xy*z;
        Q4mirx += ew.mom.xxyz*xyz;
        Q4miry += ew.mom.xyyz*xyz;
        Q4mirz += ew.mom.xyzz*xyz;

	const  momFloat zz = f_1_2*z*z;
        Q3mirx += ew.mom.xzz*zz;
        Q3miry += ew.mom.yzz*zz;
        Q3mirz += ew.mom.zzz*zz;
        const  momFloat xzz = zz*x;
        Q4mirx += ew.mom.xxzz*xzz;
        Q4miry += ew.mom.xyzz*xzz;
        Q4mirz += ew.mom.xzzz*xzz;
        const  momFloat yzz = zz*y;
        Q4mirx += ew.mom.xyzz*yzz;
        Q4miry += ew.mom.yyzz*yzz;
        Q4mirz += ew.mom.yzzz*yzz;
	const  momFloat zzz = f_1_3*zz*z;
        Q4mirx += ew.mom.xzzz*zzz;
        Q4miry += ew.mom.yzzz*zzz;
        Q4mirz += ew.mom.zzzz*zzz;

        tax += g4*Q4mirx;
        tay += g4*Q4miry;
        taz += g4*Q4mirz;
	const momFloat Q4mir = f_1_4*(Q4mirx*x + Q4miry*y + Q4mirz*z);
        dPot -= g4*Q4mir;

        const  momFloat xz = x*z;
        Q3mirx += ew.mom.xxz*xz;
        Q3miry += ew.mom.xyz*xz;
        Q3mirz += ew.mom.xzz*xz;

        const  momFloat yz = y*z;
        Q3mirx += ew.mom.xyz*yz;
        Q3miry += ew.mom.yyz*yz;
        Q3mirz += ew.mom.yzz*yz;

        const momFloat Q4x = ew.Q4xx*x + ew.Q4xy*y + ew.Q4xz*z;
        const momFloat Q4y = ew.Q4xy*x + ew.Q4yy*y + ew.Q4yz*z;
        const momFloat Q4z = ew.Q4xz*x + ew.Q4yz*y + ew.Q4zz*z;
        const momFloat Q3mir = f_1_3*(Q3mirx*x + Q3miry*y + Q3mirz*z) - 0.5*(Q4x*x + Q4y*y + Q4z*z);
        dPot -= g3*Q3mir;
        tax += g3*(Q3mirx - Q4x);
        tay += g3*(Q3miry - Q4y);
        taz += g3*(Q3mirz - Q4z);

        const momFloat Q2mirx = ew.mom.xx*x + ew.mom.xy*y + ew.mom.xz*z;
        const momFloat Q2miry = ew.mom.xy*x + ew.mom.yy*y + ew.mom.yz*z;
        const momFloat Q2mirz = ew.mom.xz*x + ew.mom.yz*y + ew.mom.zz*z;
        const momFloat Q2mir = 0.5*(Q2mirx*x + Q2miry*y + Q2mirz*z) - (ew.Q3x*x + ew.Q3y*y + ew.Q3z*z) + ew.Q4;
        dPot -= g2*Q2mir;
        tax += g2*(Q2mirx - ew.Q3x);
        tay += g2*(Q2miry - ew.Q3y);
        taz += g2*(Q2mirz - ew.Q3z);

        const momFloat Qta = g1*ew.mom.m - g2*ew.Q2 + g3*Q2mir + g4*Q3mir + g5*Q4mir;
        tax -= x*Qta;
        tay -= y*Qta;
        taz -= z*Qta;
        dFlop += COST_FLOP_EWALD;
	}

/*    dFlop += COST_FLOP_HLOOP * ew.nEwhLoop;*/ /* Accounted for outside */
    Xout[pidx] = tax;
    Yout[pidx] = tay;
    Zout[pidx] = taz;
    pPot[pidx] = dPot;
    pdFlop[pidx] = dFlop;
    }

/* If this returns an error, then the caller must attempt recovery or abort */
hipError_t cuda_setup_ewald(CUDACTX cuda) {
    if (cuda->ewIn && cuda->ewt) {
        double start = CUDA_getTime();
        CUDA_RETURN(hipMemcpyToSymbolAsync,(ew,cuda->ewIn,sizeof(ew),0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(hx,cuda->ewt->hx.f,sizeof(float)*cuda->ewIn->nEwhLoop,0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(hy,cuda->ewt->hy.f,sizeof(float)*cuda->ewIn->nEwhLoop,0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(hz,cuda->ewt->hz.f,sizeof(float)*cuda->ewIn->nEwhLoop,0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(hCfac,cuda->ewt->hCfac.f,sizeof(float)*cuda->ewIn->nEwhLoop,0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(hSfac,cuda->ewt->hSfac.f,sizeof(float)*cuda->ewIn->nEwhLoop,0,hipMemcpyHostToDevice,cuda->streamEwald));
// Time(%)      Time     Calls       Avg       Min       Max  Name
// 14.93%  1.47255s       413  3.5655ms  2.6458ms  3.9733ms  cudaEwald(double*, double*, double*, double*, double*, double*, double*, d
        momFloat dLx[MAX_TOTAL_REPLICAS];
        momFloat dLy[MAX_TOTAL_REPLICAS];
        momFloat dLz[MAX_TOTAL_REPLICAS];
        int ibHole[MAX_TOTAL_REPLICAS];
        int i=0, ix, iy, iz;
        for(ix=-3; ix<=3; ++ix) {
            for(iy=-3; iy<=3; ++iy) {
                for(iz=-3; iz<=3; ++iz) {
                    ibHole[i] = (abs(ix) <= cuda->ewIn->nReps && abs(iy) <= cuda->ewIn->nReps && abs(iz) <= cuda->ewIn->nReps);
                    dLx[i] = cuda->ewIn->Lbox * ix;
                    dLy[i] = cuda->ewIn->Lbox * iy;
                    dLz[i] = cuda->ewIn->Lbox * iz;
                    ++i;
                    }
                }
            }
        CUDA_RETURN(hipMemcpyToSymbolAsync,(Lx,dLx,sizeof(Lx),0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(Ly,dLy,sizeof(Ly),0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(Lz,dLz,sizeof(Lz),0,hipMemcpyHostToDevice,cuda->streamEwald));
        CUDA_RETURN(hipMemcpyToSymbolAsync,(bHole,ibHole,sizeof(bHole),0,hipMemcpyHostToDevice,cuda->streamEwald));


#ifdef USE_CUDA_EVENTS
        CUDA_RETURN(hipEventRecord,(cuda->eventEwald,cuda->streamEwald));
#endif
        hipError_t rc;
        do {
#ifdef USE_CUDA_EVENTS
            rc = hipEventQuery(cuda->eventEwald);
#else
            rc = hipStreamQuery(cuda->streamEwald);
#endif
            switch(rc) {
            case hipSuccess:
            case hipErrorNotReady:
                break;
            default:
                return rc;
                }
            if (CUDA_getTime() - start > 1.0) {
                return hipErrorLaunchTimeOut;
                }
            } while (rc!=hipSuccess);
        }
    return hipSuccess;
    }

extern "C"
void cudaEwaldInit(void *cudaCtx, struct EwaldVariables *ewIn, EwaldTable *ewt ) {
    CUDACTX cuda = reinterpret_cast<CUDACTX>(cudaCtx);
    cuda->ewIn = ewIn;
    cuda->ewt = ewt;
    if (cuda->iCore==0) {
        hipError_t ec = cuda_setup_ewald(cuda);
        if (ec != hipSuccess) CUDA_attempt_recovery(cuda,ec);
        }
    }

/* If this returns an error, then the caller must attempt recovery or abort */
extern "C"
int CUDAinitWorkEwald( void *ve, void *vwork ) {
    workEwald *e = reinterpret_cast<workEwald *>(ve);
    CUDAwqNode *work = reinterpret_cast<CUDAwqNode *>(vwork);
    momFloat *pHostBufFromGPU  = reinterpret_cast<momFloat *>(work->pHostBufFromGPU);
    momFloat *pHostBufToGPU    = reinterpret_cast<momFloat *>(work->pHostBufToGPU);
    momFloat *pCudaBufIn = reinterpret_cast<momFloat *>(work->pCudaBufIn);
    momFloat *pCudaBufOut = reinterpret_cast<momFloat *>(work->pCudaBufOut);
    momFloat *X, *Y, *Z;
    momFloat *cudaX, *cudaY, *cudaZ, *cudaXout, *cudaYout, *cudaZout, *cudaPot, *cudaFlop;
    int align, i;

    align = (e->nP+MASK)&~MASK; /* Warp align the memory buffers */
    X       = pHostBufToGPU + 0*align;
    Y       = pHostBufToGPU + 1*align;
    Z       = pHostBufToGPU + 2*align;
    cudaX   = pCudaBufIn + 0*align;
    cudaY   = pCudaBufIn + 1*align;
    cudaZ   = pCudaBufIn + 2*align;
    cudaXout= pCudaBufOut + 0*align;
    cudaYout= pCudaBufOut + 1*align;
    cudaZout= pCudaBufOut + 2*align;
    cudaPot = pCudaBufOut + 3*align;
    cudaFlop= pCudaBufOut + 4*align;

    dim3 dimBlock( ALIGN, 1 );
    dim3 dimGrid( align/ALIGN, 1,1 );
    for(i=0; i<e->nP; ++i) {
        const workParticle *wp = e->ppWorkPart[i];
	const int wi = e->piWorkPart[i];
	const PINFOIN *in = &wp->pInfoIn[wi];
	X[i] = wp->c[0] + in->r[0];
	Y[i] = wp->c[1] + in->r[1];
	Z[i] = wp->c[2] + in->r[2];
	}
    for(;i<align;++i) X[i]=Y[i]=Z[i] = 100;

    // copy data directly to device memory
    CUDA_RETURN(hipMemcpyAsync,(pCudaBufIn, pHostBufToGPU, align*3*sizeof(momFloat),
	    hipMemcpyHostToDevice, work->stream));
    cudaEwald<<<dimGrid, dimBlock, 0, work->stream>>>(cudaX,cudaY,cudaZ,cudaXout,cudaYout,cudaZout,cudaPot,cudaFlop);
    CUDA_RETURN(hipMemcpyAsync,(pHostBufFromGPU, pCudaBufOut, align*5*sizeof(momFloat),
            hipMemcpyDeviceToHost, work->stream));
#ifdef USE_CUDA_EVENTS
    CUDA_RETURN(hipEventRecord,(work->event,work->stream));
#endif

    return hipSuccess;
    }

extern "C"
void pkdAccumulateCUDA(void * pkd,workEwald *we,momFloat *pax,momFloat *pay,momFloat *paz,momFloat *pot,momFloat *pdFlop);


extern "C"
int CUDAcheckWorkEwald( void *ve, void *vwork ) {
    workEwald *e = reinterpret_cast<workEwald *>(ve);
    CUDAwqNode *work = reinterpret_cast<CUDAwqNode *>(vwork);
    momFloat *pHostBuf = reinterpret_cast<momFloat *>(work->pHostBufFromGPU);
    momFloat *X, *Y, *Z, *pPot, *pdFlop;
    int align;

    align = (e->nP+MASK)&~MASK; /* As above! Warp align the memory buffers */
    X       = pHostBuf + 0*align;
    Y       = pHostBuf + 1*align;
    Z       = pHostBuf + 2*align;
    pPot    = pHostBuf + 3*align;
    pdFlop  = pHostBuf + 4*align;
    pkdAccumulateCUDA(e->pkd,e,X,Y,Z,pPot,pdFlop);
    free(e->ppWorkPart);
    free(e->piWorkPart);
    free(e);
    return 0;

    }
