#include "mdlcuda.h"
#include <stdio.h>
#include <assert.h>
#include <algorithm>

namespace mdl {

#define CUDA_CHECK(f,a) {hipError_t rc = (f)a; if (rc!=hipSuccess) Abort(rc,#f,__FILE__,__LINE__);}
#define CUDA_RETURN(f,a) {hipError_t rc = (f)a; if (rc!=hipSuccess) return rc;}

void Abort(hipError_t rc, const char *fname, const char *file, int line) {
    fprintf(stderr,"%s error %d in %s(%d)\n%s\n", fname, rc, file, line, hipGetErrorString(rc));
    exit(1);
    }

/*****************************************************************************\
* CUDA : CUDA devices manager
\*****************************************************************************/

CUDA::CUDA() : pDevice(nullptr), nDevices(0) {
}

CUDA::~CUDA() {
    if (pDevice) delete [] pDevice;
}

// Create a set of device objects, each with "n" streams
void CUDA::initialize(int nStreamsPerDevice) {
    if (hipGetDeviceCount(&nDevices) != hipSuccess) nDevices = 0;
    pDevice = new Device*[nDevices];
    for (auto iDevice=0; iDevice<nDevices; ++iDevice) {
        devices.emplace_back(iDevice,nStreamsPerDevice);
        pDevice[iDevice] = &devices.back();
    }
}

// If we can start some work then do so.
void CUDA::initiate() {
    while (!empty()) { // A message is waiting. Find a stream if we can.
        assert(devices.size()>0); // This would be odd at this point. No progress could be made.
        // Find the device with the fewest busy streams (most idle streams)
        auto device = std::min_element(devices.begin(),devices.end(),Device::compareBusy);
        if (device != devices.end() && !device->empty()) {
            cudaMessage &M = dequeue();
            auto iDevice = M.getDevice();
            if (iDevice<0) device->launch(M); // Launch message M on the most idle device.
            else pDevice[iDevice]->launch(M); // Otherwise launch on the specified device.
        }
        else break; // No free streams to launch work at this time
    }
}

/*****************************************************************************\
* Device : Control for a single device
\*****************************************************************************/

Device::Device(int iDevice, int nStreams) : iDevice(iDevice), nStreams(nStreams), busy_streams(0) {
    for(auto i=0; i<nStreams; ++i) {
    	free_streams.enqueue(new Stream(this));
	}
    }

void Device::launch(cudaMessage &M) {
    if (free_streams.empty()) abort();
    auto &stream = free_streams.dequeue();
    auto stm = stream.getStream(); // the CUDA stream (hipSetDevice is called)
    stream.message = &M; // Save the message (for kernel_finished)
    ++busy_streams; // This is atomic
    M.launch(stm,stream.pCudaBufIn,stream.pCudaBufOut); // message specific launch operation
    // Ask CUDA to notify us when the prior queued work has finished
    hipLaunchHostFunc(stm,Device::kernel_finished,&stream);
    }

// Static "void *" version: recover the Stream object and call
void CUDART_CB Device::kernel_finished( void*  userData ) {
    auto stream = reinterpret_cast<Stream *>(userData);
    stream->device->kernel_finished(stream);
    }

// Here we move the Stream back to the free list and return the message
// to the requester. CAREFUL: this is called from a special CUDA thread.
void Device::kernel_finished( Stream *stream ) {
    stream->message->sendBack();
    stream->message = NULL;
    --busy_streams; // This is atomic
    free_streams.enqueue(stream);
    }

/*****************************************************************************\
* Stream : a stream on a specific device
\*****************************************************************************/

Stream::Stream(class Device *device) : device(device), message(0) {
    CUDA_CHECK(hipSetDevice,(device->iDevice)); // Stream is for this device
    CUDA_CHECK(hipStreamCreate, (&stream));     // CUDA stream
    CUDA_CHECK(hipMalloc,(&pCudaBufIn,  requestBufferSize));
    CUDA_CHECK(hipMalloc,(&pCudaBufOut, resultsBufferSize));
    }

// Destroy the stream on the correct device.
Stream::~Stream() {
    CUDA_CHECK(hipSetDevice,(device->iDevice));
    CUDA_CHECK(hipFree,(pCudaBufIn));
    CUDA_CHECK(hipFree,(pCudaBufOut));
    hipStreamDestroy(stream);
    }

// Activate the appropriate device, and return the stream
hipStream_t Stream::getStream() {
    CUDA_CHECK(hipSetDevice,(device->iDevice));
    return stream;
}

} // namespace mdl
